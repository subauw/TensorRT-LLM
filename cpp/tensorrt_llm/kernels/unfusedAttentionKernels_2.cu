#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 * Copyright (c) 2021, NAVER Corp.  Authored by CLOVA.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

// Separate from unfusedAttentionKernel to accelerate compiling.

#include "tensorrt_llm/common/assert.h"
#include "tensorrt_llm/common/cudaTypeUtils.cuh"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/decoderMaskedMultiheadAttentionUtils.h"
#include "tensorrt_llm/kernels/gptKernels.h"
#include "tensorrt_llm/kernels/kvCacheUtils.h"
#include "tensorrt_llm/kernels/unfusedAttentionKernels.h"

using namespace tensorrt_llm::common;

namespace tensorrt_llm
{
namespace kernels
{

template <typename T>
struct Vec_t
{
    static constexpr int size = 0;
};

template <>
struct Vec_t<float>
{
    using Type = float2;
    static constexpr int size = 2;
};

template <>
struct Vec_t<half>
{
    using Type = uint32_t;
    static constexpr int size = 2;
};

#ifdef ENABLE_BF16
template <>
struct Vec_t<__hip_bfloat16>
{
    using Type = __hip_bfloat162;
    static constexpr int size = 2;
};
#endif

template <typename T, typename T_cache, bool ADD_BIAS, bool STORE_QKV, typename KVCacheBuffer, bool IsGenerate>
__global__ void applyBiasRopeUpdateKVCache(T* QKV, T* Q, KVCacheBuffer kvCacheBuffer, const T* __restrict qkv_bias,
    const int* seq_lens, const int* kv_seq_lens, const int* padding_offset, const float* kvScaleOrigQuant,
    const int batch_size, const int seq_len, const int cyclic_kv_cache_len, const int head_num, const int kv_head_num,
    const int size_per_head, const int rotary_embedding_dim, float rotary_embedding_base,
    RotaryScalingType const rotary_scale_type, float rotary_embedding_scale, const int rotary_embedding_max_positions,
    PositionEmbeddingType const position_embedding_type, int beam_width)
{
    // This kernel add bias to QKV, which has shape [batch_size, seq_len, 3, head_num, size_per_head], and
    // QKV split to 3 split buffer q, k, v and transpose them to [batch_size, head_num, seq_len, size_per_head].
    // For q and k, also apply the rotary embedding.

    // NOTE:
    // head_num == kv_head_num
    //   QKV src shape (batch_size, seq_len, 3, head_num, size_per_head)
    //                  ^^^^^^^^^^^^^^^^^^^  ^^^^^^^^^^^^^^^^^^^^^^^^^^
    //                           m                        n
    //   QKV dst shape (3, batch_size, head_num, seq_len, size_per_head)
    // head_num != kv_head_num
    //   QKV src shape: (batch_size, seq_len, head_num * size_per_head + 2 * kv_head_num * size_per_head)
    //                   ^^^^^^^^^^^^^^^^^^^  ^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^
    //                             m                               n
    //   Q dst shape: (batch_size, head_num, seq_len, size_per_head)
    //   KV dst shape: (batch_size, kv_head_num, seq_len, size_per_head)

    // FMHA with paged kv cache input:
    // Need separate contiguous Q buffer.

    extern __shared__ __align__(sizeof(float2)) char smem_[]; // align on largest vector type

    constexpr int vec_size = Vec_t<T>::size;
    using Vec_t = typename Vec_t<T>::Type;
    const int token_idx = blockIdx.x;
    const bool has_padding = padding_offset == nullptr;

    constexpr bool ENABLE_8BITS_CACHE = sizeof(T_cache) == 1;
    constexpr int X_ELEMS = vec_size;
    const int sizePerHeadDivX = size_per_head / X_ELEMS;
    using T_dst = T_cache;

    // The index of the token in the batch. It includes "virtual" padding (even if the input is not padded)
    // such that the sequence index and the position in the sequence can be obtained using the max.
    // sequence length as:
    const int token_padding_offset = (has_padding || IsGenerate) ? 0 : padding_offset[token_idx];
    const int global_token_idx = (!IsGenerate) ? token_idx + token_padding_offset : token_idx;
    const int batch_beam_idx = global_token_idx / seq_len;
    const int batch_idx = (!IsGenerate) ? batch_beam_idx : batch_beam_idx / beam_width;
    const int final_kv_seq_len = (!IsGenerate) ? kv_seq_lens[batch_idx] : 0;
    const int actual_seq_len = seq_lens[batch_idx];
    const int token_idx_in_seq
        = (!IsGenerate) ? (final_kv_seq_len - actual_seq_len) + global_token_idx % seq_len : actual_seq_len - 1;
    const bool valid_seq = IsGenerate || (token_idx_in_seq < actual_seq_len || !has_padding);

    const int head_idx = blockIdx.y;
    const int tidx = threadIdx.x;

    const bool is_seq_masked = !valid_seq;
    const bool is_head_size_masked = tidx * vec_size >= size_per_head;
    const bool is_masked = is_head_size_masked || is_seq_masked;

    const int hidden_size = head_num * size_per_head;
    const int hidden_idx = head_idx * size_per_head + tidx * vec_size;
    const int qheads_per_kv_head = head_num / kv_head_num;
    const int kv_head_idx = head_idx / qheads_per_kv_head;
    const int hidden_idx_kv = kv_head_idx * size_per_head + tidx * vec_size;
    const int n = (head_num + 2 * kv_head_num) * size_per_head;

    const int dst_kv_seq_idx = token_idx_in_seq;
    const int src_k_offset = hidden_size;
    const int src_v_offset = hidden_size + kv_head_num * size_per_head;

    // NOTE: q has seq len excluding prefix prompt
    // head_num == kv_head_num:
    //   src QKV: [batch, time, 3, head_num, size_per_head]
    // head_num != kv_head_num:
    //   src QKV: [batch, time, head_num * size_per_head + 2 * kv_head_num * size_per_head]
    const int src_q_idx = token_idx * n + hidden_idx;
    const int src_k_idx = token_idx * n + src_k_offset + hidden_idx_kv;
    const int src_v_idx = token_idx * n + src_v_offset + hidden_idx_kv;

    Vec_t q, k, v, zero;
    Vec_t q_bias, k_bias, v_bias;
    if (valid_seq)
    {
        mmha::update_rotary_base_n_scale(rotary_embedding_base, rotary_embedding_scale, rotary_scale_type,
            rotary_embedding_dim, rotary_embedding_max_positions, actual_seq_len);
    }

#pragma unroll
    for (int i = 0; i < sizeof(Vec_t) / sizeof(uint32_t); i++)
    {
        reinterpret_cast<uint32_t*>(&zero)[i] = 0u;
    }

    // load q,k,v and add bias
    if (!is_masked)
    {
        q = *reinterpret_cast<const Vec_t*>(&QKV[src_q_idx]);
        k = *reinterpret_cast<const Vec_t*>(&QKV[src_k_idx]);
        v = *reinterpret_cast<const Vec_t*>(&QKV[src_v_idx]);

        if (ADD_BIAS)
        {
            q_bias = *reinterpret_cast<const Vec_t*>(&qkv_bias[hidden_idx]);
            k_bias = *reinterpret_cast<const Vec_t*>(&qkv_bias[hidden_idx_kv + src_k_offset]);
            v_bias = *reinterpret_cast<const Vec_t*>(&qkv_bias[hidden_idx_kv + src_v_offset]);

            q = mmha::add(q, q_bias);
            k = mmha::add(k, k_bias);
            v = mmha::add(v, v_bias);
        }
    }

    switch (position_embedding_type)
    {
    case PositionEmbeddingType::kROPE_GPTJ:
    {
        mmha::apply_rotary_embedding(
            q, k, tidx, rotary_embedding_dim, rotary_embedding_base, rotary_embedding_scale, dst_kv_seq_idx);
        break;
    }
    case PositionEmbeddingType::kROPE_GPT_NEOX:
    {
        const bool do_rotary = !is_masked && vec_size * tidx < rotary_embedding_dim;

        T* q_smem = reinterpret_cast<T*>(smem_);
        T* k_smem = q_smem + rotary_embedding_dim;

        const int half_rotary_dim = rotary_embedding_dim / 2;
        const int half_idx = (tidx * vec_size) / half_rotary_dim;
        const int intra_half_idx = (tidx * vec_size) % half_rotary_dim;
        const int smem_pitch = half_rotary_dim; // TODO: adjust for bank conflicts?

        if (do_rotary)
        {
            *reinterpret_cast<Vec_t*>(q_smem + half_idx * smem_pitch + intra_half_idx) = q;
            *reinterpret_cast<Vec_t*>(k_smem + half_idx * smem_pitch + intra_half_idx) = k;
        }

        __syncthreads();

        const int transpose_idx = half_idx * (half_rotary_dim / 2) + intra_half_idx / 2;
        constexpr int tidx_factor = vec_size / 2;
        if (do_rotary)
        {
            mmha::vec_from_smem_transpose(q, q_smem, transpose_idx, smem_pitch);
            mmha::vec_from_smem_transpose(k, k_smem, transpose_idx, smem_pitch);

            mmha::apply_rotary_embedding(q, k, transpose_idx / tidx_factor, rotary_embedding_dim, rotary_embedding_base,
                rotary_embedding_scale, dst_kv_seq_idx);

            mmha::write_smem_transpose(q, q_smem, transpose_idx, smem_pitch);
            mmha::write_smem_transpose(k, k_smem, transpose_idx, smem_pitch);
        }

        __syncthreads();

        if (do_rotary)
        {
            q = *reinterpret_cast<Vec_t*>(q_smem + half_idx * smem_pitch + intra_half_idx);
            k = *reinterpret_cast<Vec_t*>(k_smem + half_idx * smem_pitch + intra_half_idx);
        }
        break;
    }
    }

    const int channelIdx{tidx};
    const bool valid_kv_cache_pos = kvCacheBuffer.data != nullptr // In KV-cache-less mode. No need to store KV values
        && token_idx_in_seq >= (actual_seq_len - cyclic_kv_cache_len);
    const int token_idx_in_kv_cache = token_idx_in_seq % cyclic_kv_cache_len;
    auto kDst = reinterpret_cast<T_dst*>(kvCacheBuffer.getKBlockPtr(batch_beam_idx, token_idx_in_kv_cache));
    auto vDst = reinterpret_cast<T_dst*>(kvCacheBuffer.getVBlockPtr(batch_beam_idx, token_idx_in_kv_cache));
    int inBlockIdx = kvCacheBuffer.getKVLocalIdx(token_idx_in_kv_cache, kv_head_idx, sizePerHeadDivX, channelIdx);
    if (!is_masked)
    {
        if constexpr (STORE_QKV)
        {
            *reinterpret_cast<Vec_t*>(&QKV[src_q_idx]) = q;
        }
        else
        {
            *reinterpret_cast<Vec_t*>(&Q[token_idx * head_num * size_per_head + hidden_idx]) = q;
        }
        if ((head_num == kv_head_num) || (head_idx == (kv_head_idx * qheads_per_kv_head)))
        {
            if constexpr (STORE_QKV)
            {
                *reinterpret_cast<Vec_t*>(&QKV[src_k_idx]) = k;
                *reinterpret_cast<Vec_t*>(&QKV[src_v_idx]) = v;
            }

            if (valid_kv_cache_pos)
            {
                if (ENABLE_8BITS_CACHE)
                {
                    inBlockIdx = inBlockIdx * vec_size;
                    // Cast float scale to dst data type.
                    using T_scale = typename mmha::kv_cache_scale_type_t<T, T_cache>::Type;
                    T_scale scaleOrigQuant;
                    mmha::convert_from_float(&scaleOrigQuant, kvScaleOrigQuant[0]);
                    // Store 8bits kv cache.
                    mmha::store_8bits_kv_cache_vec(kDst, k, inBlockIdx, scaleOrigQuant);
                    mmha::store_8bits_kv_cache_vec(vDst, v, inBlockIdx, scaleOrigQuant);
                }
                else
                {
                    reinterpret_cast<Vec_t*>(kDst)[inBlockIdx] = k;
                    reinterpret_cast<Vec_t*>(vDst)[inBlockIdx] = v;
                }
            }
        }
    }
}

#define APPLY_BIAS_ROPE_UPDATE_KV_CACHE(ADD_BIAS, STORE_QKV)                                                           \
    applyBiasRopeUpdateKVCache<T, T_cache, ADD_BIAS, STORE_QKV, KVCacheBuffer, IsGenerate>                             \
        <<<grid, block, smem_size, stream>>>(QKV, Q, kvTable, qkv_bias, seq_lens, kv_seq_lens, padding_offset,         \
            kvScaleOrigQuant, batch_size, seq_len, cyclic_kv_cache_len, head_num, kv_head_num, size_per_head,          \
            rotary_embedding_dim, rotary_embedding_base, rotary_scale_type, rotary_embedding_scale,                    \
            rotary_embedding_max_positions, position_embedding_type, beam_width);

template <typename T, typename T_cache, typename KVCacheBuffer, bool IsGenerate>
void invokeApplyBiasRopeUpdateKVCacheDispatch(T* QKV, T* Q, KVCacheBuffer& kvTable, const T* qkv_bias,
    const int* seq_lens, const int* kv_seq_lens, const int* padding_offset, const int batch_size, const int seq_len,
    const int cyclic_kv_cache_len, const int token_num, const int head_num, const int kv_head_num,
    const int size_per_head, const int rotary_embedding_dim, const float rotary_embedding_base,
    const RotaryScalingType rotary_scale_type, const float rotary_embedding_scale,
    const int rotary_embedding_max_positions, const PositionEmbeddingType position_embedding_type, const float* scale,
    const float* kvScaleOrigQuant, const int int8_mode, const bool enable_paged_kv_fmha, hipStream_t stream,
    int beam_width)
{
    TLLM_CHECK_WITH_INFO(int8_mode != 2, "w8a8 not yet implemented with RoPE"); // TODO
    if constexpr (!IsGenerate)
    {
        TLLM_CHECK_WITH_INFO(beam_width == 1, "beam_width should be default 1 for context phase.");
    }
    else
    {
        TLLM_CHECK_WITH_INFO(seq_len == 1, "Generation phase should have seq_len of 1.");
        TLLM_CHECK_WITH_INFO(padding_offset == nullptr, "Generation phase should not use padding_offset");
        TLLM_CHECK_WITH_INFO(
            token_num == batch_size * beam_width, "token_num should be batch_size * beam_width for generation phase.");
    }
    // To implement rotary embeddings, each thread processes two QKV elems:
    dim3 block((size_per_head / Vec_t<T>::size + 31) / 32 * 32);
    dim3 grid(token_num, head_num);
    size_t smem_size
        = (position_embedding_type == PositionEmbeddingType::kROPE_GPT_NEOX ? 2 * rotary_embedding_dim * sizeof(T) : 0);

    // Launch template parameters.
    const bool add_bias = qkv_bias != nullptr;
    const bool store_qkv = !enable_paged_kv_fmha;

    // NOTE: add offset for rotary embedding
    if (add_bias)
    {
        if (store_qkv)
        {
            APPLY_BIAS_ROPE_UPDATE_KV_CACHE(true, true);
        }
        else
        {
            APPLY_BIAS_ROPE_UPDATE_KV_CACHE(true, false);
        }
    }
    else
    {
        if (store_qkv)
        {
            APPLY_BIAS_ROPE_UPDATE_KV_CACHE(false, true);
        }
        else
        {
            APPLY_BIAS_ROPE_UPDATE_KV_CACHE(false, false);
        }
    }
}

template <typename T, typename KVCacheBuffer, bool IsGenerate>
void invokeApplyBiasRopeUpdateKVCache(T* QKV, T* Q, KVCacheBuffer& kvTable, const T* qkv_bias, const int* seq_lens,
    const int* kv_seq_lens, const int* padding_offset, const int batch_size, const int seq_len,
    const int cyclic_kv_cache_len, const int token_num, const int head_num, const int kv_head_num,
    const int size_per_head, const int rotary_embedding_dim, const float rotary_embedding_base,
    const RotaryScalingType rotary_scale_type, const float rotary_embedding_scale,
    const int rotary_embedding_max_positions, const PositionEmbeddingType position_embedding_type, const float* scale,
    const int int8_mode, const KvCacheDataType cache_type, const float* kvScaleOrigQuant,
    const bool enable_paged_kv_fmha, hipStream_t stream, int beam_width)
{
    // Block handles both K and V tile.
    constexpr int x = (sizeof(T) == 4) ? 4 : 8;
    TLLM_CHECK_WITH_INFO(size_per_head % x == 0, "Size per head is not a multiple of X");

    if (cache_type == KvCacheDataType::INT8)
    {
        invokeApplyBiasRopeUpdateKVCacheDispatch<T, int8_t, KVCacheBuffer, IsGenerate>(QKV, Q, kvTable, qkv_bias,
            seq_lens, kv_seq_lens, padding_offset, batch_size, seq_len, cyclic_kv_cache_len, token_num, head_num,
            kv_head_num, size_per_head, rotary_embedding_dim, rotary_embedding_base, rotary_scale_type,
            rotary_embedding_scale, rotary_embedding_max_positions, position_embedding_type, scale, kvScaleOrigQuant,
            int8_mode, enable_paged_kv_fmha, stream, beam_width);
    }
#ifdef ENABLE_FP8
    else if (cache_type == KvCacheDataType::FP8)
    {
        invokeApplyBiasRopeUpdateKVCacheDispatch<T, __hip_fp8_e4m3_fnuz, KVCacheBuffer, IsGenerate>(QKV, Q, kvTable, qkv_bias,
            seq_lens, kv_seq_lens, padding_offset, batch_size, seq_len, cyclic_kv_cache_len, token_num, head_num,
            kv_head_num, size_per_head, rotary_embedding_dim, rotary_embedding_base, rotary_scale_type,
            rotary_embedding_scale, rotary_embedding_max_positions, position_embedding_type, scale, kvScaleOrigQuant,
            int8_mode, enable_paged_kv_fmha, stream, beam_width);
    }
#endif // ENABLE_FP8
    else
    {
        invokeApplyBiasRopeUpdateKVCacheDispatch<T, T, KVCacheBuffer, IsGenerate>(QKV, Q, kvTable, qkv_bias, seq_lens,
            kv_seq_lens, padding_offset, batch_size, seq_len, cyclic_kv_cache_len, token_num, head_num, kv_head_num,
            size_per_head, rotary_embedding_dim, rotary_embedding_base, rotary_scale_type, rotary_embedding_scale,
            rotary_embedding_max_positions, position_embedding_type, scale, kvScaleOrigQuant, int8_mode,
            enable_paged_kv_fmha, stream, beam_width);
    }
}

#define INSTANTIATE_ADDFUSEDQKVBIAS_TRANSPOSE(T, KVCacheBuffer, IsGenerate)                                            \
    template void invokeApplyBiasRopeUpdateKVCache<T, KVCacheBuffer, IsGenerate>(T * QKV, T * Q,                       \
        KVCacheBuffer & kvTable, const T* qkv_bias, const int* seq_lens, const int* kv_seq_lens,                       \
        const int* padding_offset, const int batch_size, const int seq_len, const int cyclic_kv_cache_len,             \
        const int token_num, const int head_num, const int kv_head_num, const int size_per_head,                       \
        const int rotary_embedding_dim, const float rotary_embedding_base, const RotaryScalingType rotary_scale_type,  \
        const float rotary_embedding_scale, const int rotary_embedding_max_positions,                                  \
        const PositionEmbeddingType position_embedding_type, const float* scale, const int int8_mode,                  \
        const KvCacheDataType cache_type, const float* kvScaleOrigQuant, const bool enable_paged_kv_fmha,              \
        hipStream_t stream, int beam_width)

INSTANTIATE_ADDFUSEDQKVBIAS_TRANSPOSE(float, KVBlockArray, false);
INSTANTIATE_ADDFUSEDQKVBIAS_TRANSPOSE(float, KVLinearBuffer, false);
INSTANTIATE_ADDFUSEDQKVBIAS_TRANSPOSE(half, KVBlockArray, false);
INSTANTIATE_ADDFUSEDQKVBIAS_TRANSPOSE(half, KVLinearBuffer, false);
INSTANTIATE_ADDFUSEDQKVBIAS_TRANSPOSE(float, KVBlockArray, true);
INSTANTIATE_ADDFUSEDQKVBIAS_TRANSPOSE(float, KVLinearBuffer, true);
INSTANTIATE_ADDFUSEDQKVBIAS_TRANSPOSE(half, KVBlockArray, true);
INSTANTIATE_ADDFUSEDQKVBIAS_TRANSPOSE(half, KVLinearBuffer, true);
#ifdef ENABLE_BF16
INSTANTIATE_ADDFUSEDQKVBIAS_TRANSPOSE(__hip_bfloat16, KVBlockArray, false);
INSTANTIATE_ADDFUSEDQKVBIAS_TRANSPOSE(__hip_bfloat16, KVLinearBuffer, false);
INSTANTIATE_ADDFUSEDQKVBIAS_TRANSPOSE(__hip_bfloat16, KVBlockArray, true);
INSTANTIATE_ADDFUSEDQKVBIAS_TRANSPOSE(__hip_bfloat16, KVLinearBuffer, true);
#endif
#undef INSTANTIATE_ADDFUSEDQKVBIAS_TRANSPOSE

} // namespace kernels
} // namespace tensorrt_llm
