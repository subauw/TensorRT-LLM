#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/kernels/beamSearchPenaltyKernels.h"
#include "tensorrt_llm/layers/baseBeamSearchLayer.h"
#include "tensorrt_llm/layers/fillBuffers.h"

#include <algorithm>

using namespace tensorrt_llm::common;
using namespace tensorrt_llm::kernels;

namespace tensorrt_llm
{
namespace layers
{

__global__ void update_indir_cache_kernel(int* tgt_indir_cache, const int* src_indir_cache, const int** parent_ids,
    const FinishedState* finished, const int* sequence_lengths, const int* input_lengths, int batch_dim,
    int local_batch_size, int beam_width, int max_attention_window, int max_seq_len)
{
    int time_step = threadIdx.x + blockIdx.x * blockDim.x;
    int bb_id = threadIdx.y + blockIdx.y * blockDim.y;   // should be just blockIdx.y?
    const int current_step{sequence_lengths[bb_id] - 1}; // the sequence_lengths is updated, need to minus 1
    const int input_length{input_lengths == nullptr ? 0 : input_lengths[bb_id]};
    const int batch_id = bb_id / beam_width;
    const int beam_id = bb_id % beam_width;
    // Exit when the batch_beam or timestep is out of the bound.
    // Assume that KV Cache is shared and fixed for context part,
    //  so we don't need to update the indices for context part.
    if (bb_id >= beam_width * local_batch_size || time_step >= max_seq_len || time_step < input_length
        || time_step < (max_seq_len - max_attention_window) || finished[bb_id].isFinished())
    {
        return;
    }
    int time_step_circ = time_step % max_attention_window;

    // for the parent_ids, we will still keep it for all past tokens (i.e. max_seq_len)
    const int src_beam = parent_ids[batch_id][beam_id * max_seq_len + current_step];

    // for the indir tables, we have the cyclic kv cache.
    const uint32_t tgt_offset
        = batch_id * beam_width * max_attention_window + beam_id * max_attention_window + time_step_circ;
    const uint32_t src_offset
        = batch_id * beam_width * max_attention_window + src_beam * max_attention_window + time_step_circ;

    tgt_indir_cache[tgt_offset] = (time_step == current_step) ? beam_id : src_indir_cache[src_offset];
}

void update_indir_cache_kernelLauncher(int* tgt_indir_cache, const int* src_indir_cache, const int** parent_ids,
    const FinishedState* finished, const int* sequence_lengths, const int* input_lengths, int batch_dim,
    int local_batch_size, int beam_width, int max_seq_len, int max_attention_window, hipStream_t stream)
{
    const dim3 block(32);
    // Update indirections steps [input_length[bb_id], sequence_lengths[bb_id]], included
    const dim3 grid((max_seq_len + block.x - 1) / block.x, local_batch_size * beam_width);
    update_indir_cache_kernel<<<grid, block, 0, stream>>>(tgt_indir_cache, src_indir_cache, parent_ids, finished,
        sequence_lengths, input_lengths, batch_dim, local_batch_size, beam_width, max_attention_window, max_seq_len);
}

template <typename T>
BaseBeamSearchLayer<T>::BaseBeamSearchLayer(size_t vocab_size, size_t vocab_size_padded, hipStream_t stream,
    std::shared_ptr<IAllocator> allocator, bool is_free_buffer_after_forward)
    : BaseLayer(stream, std::move(allocator), is_free_buffer_after_forward, nullptr)
    , vocab_size_(vocab_size)
    , vocab_size_padded_(vocab_size_padded)
{
}

template <typename T>
BaseBeamSearchLayer<T>::BaseBeamSearchLayer(BaseBeamSearchLayer<T> const& beam_search_layer)
    : BaseLayer(beam_search_layer)
    , vocab_size_(beam_search_layer.vocab_size_)
    , vocab_size_padded_(beam_search_layer.vocab_size_padded_)
    , topk_softmax_workspace_size_(beam_search_layer.topk_softmax_workspace_size_)
{
}

template <typename T>
BaseBeamSearchLayer<T>::~BaseBeamSearchLayer()
{
    TLLM_LOG_TRACE(__PRETTY_FUNCTION__);
    freeBuffer();
}

template <typename T>
void BaseBeamSearchLayer<T>::freeBuffer()
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);
    if (is_allocate_buffer_)
    {
        allocator_->free((void**) (&temperature_buf_));
        allocator_->free((void**) (&min_lengths_buf_));
        allocator_->free((void**) (&repetition_penalty_buf_));
        allocator_->free((void**) (&presence_penalty_buf_));
        allocator_->free((void**) (&frequency_penalty_buf_));
        is_allocate_buffer_ = false;
    }
    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void BaseBeamSearchLayer<T>::allocateBuffer(size_t batch_size)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);
    temperature_buf_ = allocator_->reMalloc(temperature_buf_, sizeof(float) * batch_size, false);
    min_lengths_buf_ = allocator_->reMalloc(min_lengths_buf_, sizeof(int) * batch_size, false);
    repetition_penalty_buf_ = allocator_->reMalloc(repetition_penalty_buf_, sizeof(float) * batch_size, false);
    presence_penalty_buf_ = allocator_->reMalloc(presence_penalty_buf_, sizeof(float) * batch_size, false);
    frequency_penalty_buf_ = allocator_->reMalloc(frequency_penalty_buf_, sizeof(float) * batch_size, false);

    is_allocate_buffer_ = true;
    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void BaseBeamSearchLayer<T>::setupBase(size_t batch_size, SetupParams const& setupParams)
{
    allocateBuffer(batch_size);
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);
    // Setup penalties.
    FillBuffers const fillBuffers{batch_size, stream_};

    fillBuffers(setupParams.temperature, 1.0f, mTemperature, temperature_buf_);
    fillBuffers(setupParams.min_length, 1, mMinLength, min_lengths_buf_);

    use_repetition_penalty_ = static_cast<bool>(setupParams.repetition_penalty);
    use_presence_penalty_ = static_cast<bool>(setupParams.presence_penalty);
    use_frequency_penalty_ = static_cast<bool>(setupParams.frequency_penalty);
    if (use_repetition_penalty_)
    {
        fillBuffers(setupParams.repetition_penalty, getDefaultPenaltyValue(RepetitionPenaltyType::Repetition),
            mRepetitionPenalty, repetition_penalty_buf_);
    }
    if (use_presence_penalty_)
    {
        fillBuffers(setupParams.presence_penalty, getDefaultPenaltyValue(RepetitionPenaltyType::Presence),
            mPresencePenalty, presence_penalty_buf_);
    }
    if (use_frequency_penalty_)
    {
        fillBuffers(setupParams.frequency_penalty, getDefaultPenaltyValue(RepetitionPenaltyType::Frequency),
            mFrequencyPenalty, frequency_penalty_buf_);
    }
    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void BaseBeamSearchLayer<T>::forward(BeamSearchOutputParams& outputs, ForwardParams const& params)
{
    TLLM_LOG_TRACE("%s", __PRETTY_FUNCTION__);
    Tensor& output_ids_ptr = outputs.output_ids_ptr;

    const auto batch_size = static_cast<std::int32_t>(output_ids_ptr.shape[0]);
    const auto beam_width = static_cast<std::int32_t>(output_ids_ptr.shape[1]);
    const auto max_seq_len = static_cast<std::int32_t>(output_ids_ptr.shape[2]);

    TLLM_CHECK_WITH_INFO(params.ite == 0, "Pipeline Parallelism is not supported yet !");

    const int ite{params.ite};
    Tensor const& logits = params.logits;
    const auto local_batch_size = logits.shape[0];

    const T* embedding_bias = params.embedding_bias ? params.embedding_bias->template getPtr<const T>() : nullptr;

    auto* end_ids = params.end_ids.template getPtr<const int>();
    auto* const input_lengths = params.input_lengths ? params.input_lengths->template getPtr<const int>() : nullptr;
    int* sequence_length = (outputs.sequence_length) ? outputs.sequence_length->template getPtr<int>() : nullptr;

    invokeAddBiasApplyPenalties(logits.getPtr<T>(), output_ids_ptr.template getPtr<const int*>(),
        outputs.parent_ids_ptr.template getPtr<const int*>(), input_lengths, sequence_length, embedding_bias, ite,
        local_batch_size, batch_size, beam_width, vocab_size_, vocab_size_padded_, end_ids, temperature_buf_,
        mTemperature, repetition_penalty_buf_, presence_penalty_buf_, frequency_penalty_buf_, mRepetitionPenalty,
        mPresencePenalty, mFrequencyPenalty, use_repetition_penalty_, use_presence_penalty_, use_frequency_penalty_,
        min_lengths_buf_, max_seq_len, stream_);
    sync_check_cuda_error();

    invokeSoftMax(outputs, params);

    if (beam_width > 1)
    {
        update_indir_cache_kernelLauncher(outputs.tgt_cache_indirection.template getPtr<int>(),
            params.src_cache_indirection.template getPtr<const int>(),
            outputs.parent_ids_ptr.template getPtr<const int*>(),
            reinterpret_cast<const FinishedState*>(
                outputs.finished->template getPtr<const FinishedState::UnderlyingType>()),
            sequence_length, input_lengths, batch_size, local_batch_size, beam_width, max_seq_len,
            params.max_attention_window, stream_);
        sync_check_cuda_error();
    }
    sync_check_cuda_error();
    if (is_free_buffer_after_forward_)
    {
        freeBuffer();
    }
    sync_check_cuda_error();
}

template class BaseBeamSearchLayer<float>;
template class BaseBeamSearchLayer<half>;

} // namespace layers
} // namespace tensorrt_llm
