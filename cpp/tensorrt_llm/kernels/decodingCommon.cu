#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/decodingCommon.h"
#include <stdio.h>

using namespace tensorrt_llm::common;

namespace tensorrt_llm
{
namespace kernels
{

__global__ void curandInitialize(hiprandState_t* state, const int size, const uint64_t randomSeed)
{
    if (threadIdx.x + blockIdx.x * blockDim.x < size)
    {
        hiprand_init(randomSeed, 0, 0, &state[blockIdx.x * blockDim.x + threadIdx.x]);
    }
}

void invokeCurandInitialize(
    hiprandState_t* state, const size_t batchSize, const uint64_t randomSeed, hipStream_t stream)
{
    dim3 block(256);
    dim3 grid((int) (ceil(batchSize * 1.0 / 256)));
    curandInitialize<<<grid, block, 0, stream>>>(state, batchSize, randomSeed);
}

__global__ void curandBatchInitialize(hiprandState_t* states, const int size, const uint64_t* randomSeeds)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size)
    {
        hiprand_init(randomSeeds[idx], 0, 0, &states[idx]);
    }
}

void invokeCurandBatchInitialize(
    hiprandState_t* states, const size_t batchSize, const uint64_t* randomSeeds, hipStream_t stream)
{
    dim3 block(256);
    dim3 grid((int) (ceil(batchSize * 1.0 / 256)));
    curandBatchInitialize<<<grid, block, 0, stream>>>(states, batchSize, randomSeeds);
}

template <typename T>
__global__ void addBiasSoftMax(T* logits, T* probs, const T* bias, const int* endIds, const FinishedState* finished,
    const int vocabSize, const int vocabSizePadded)
{
    int bid = blockIdx.x;
    const FinishedState finishState = finished != nullptr ? finished[bid] : FinishedState::empty();
    if (finishState.isSkipDecoding())
    {
        return;
    }

    bool finish = finishState.isFinished();
    int offset = bid * vocabSizePadded;

    float maxVal = -1 * FLT_MAX;
    const bool IS_FP16 = std::is_same<T, half>::value;
    const T MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : FLT_MAX;
    __shared__ float sMaxVal;
    __shared__ float sSumVal;

    for (int tid = threadIdx.x; tid < vocabSizePadded; tid += blockDim.x)
    {
        if (tid < vocabSize)
        {
            if (finish && endIds != nullptr)
            {
                logits[offset + tid] = (tid == endIds[bid]) ? MAX_T_VAL : -MAX_T_VAL;
            }
            else
            {
                T bias_val = (bias != nullptr) ? bias[tid] : (T) 0.0f;
                logits[offset + tid] += bias_val;
            }
        }
        else
        {
            logits[offset + tid] = -MAX_T_VAL;
        }
        maxVal = max(maxVal, (float) logits[offset + tid]);
    }

    maxVal = blockReduceMax<float>((float) maxVal);
    if (threadIdx.x == 0)
    {
        sMaxVal = maxVal;
    }
    __syncthreads();

    float sumVal = 0.0f;
    for (int tid = threadIdx.x; tid < vocabSizePadded; tid += blockDim.x)
    {
        probs[offset + tid] = __expf((float) logits[offset + tid] - sMaxVal);
        sumVal += (float) probs[offset + tid];
    }

    sumVal = blockReduceSum<float>(sumVal);
    if (threadIdx.x == 0)
    {
        sSumVal = sumVal;
    }
    __syncthreads();

    for (int tid = threadIdx.x; tid < vocabSizePadded; tid += blockDim.x)
    {
        probs[offset + tid] = ((float) probs[offset + tid] / (sSumVal + 1e-6f));
    }
}

template <typename T>
void invokeAddBiasSoftMax(T* logits, T* probs, const T* bias, const int* endIds, const FinishedState* finished,
    const int batchSize, const int vocabSize, const int vocabSizePadded, hipStream_t stream)
{
    dim3 grid(batchSize);
    auto const vocabRoundedToWarp = roundUp(vocabSize, 32);
    dim3 block(min(vocabRoundedToWarp, 1024));
    // vocabSize, e.g., 30000, 7000.... vocabSize is usually very big.
    addBiasSoftMax<<<grid, block, 0, stream>>>(logits, probs, bias, endIds, finished, vocabSize, vocabSizePadded);
}

template void invokeAddBiasSoftMax(float* logits, float* probs, const float* bias, const int* endIds,
    const FinishedState* finished, const int m, const int nPadded, const int n, hipStream_t stream);

template void invokeAddBiasSoftMax(half* logits, half* probs, const half* bias, const int* endIds,
    const FinishedState* finished, const int m, const int nPadded, const int n, hipStream_t stream);

} // namespace kernels
} // namespace tensorrt_llm
