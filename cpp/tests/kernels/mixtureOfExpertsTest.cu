#include "hip/hip_runtime.h"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/runtime/cudaStream.h"
#include <algorithm>
#include <gtest/gtest.h>
#include <numeric>

#include "tensorrt_llm/kernels/mixtureOfExperts/moe_kernels.h"
#include "tensorrt_llm/runtime/bufferManager.h"

using namespace tensorrt_llm::kernels;
using namespace tensorrt_llm::common;
using namespace tensorrt_llm::runtime;

template <class T>
__global__ void initWeightsKernel(T* data, int w, int h, T scalar)
{
    size_t expert_id = blockIdx.z;
    T* start_offset = data + expert_id * w * h;

    size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    size_t y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < w && y < h)
        start_offset[y * w + x] = (x == y) ? scalar : 0;
}

template <class T>
__global__ void initBiasToExpertIdKernel(T* data, int w)
{
    size_t expert_id = blockIdx.y;
    T* start_offset = data + expert_id * w;

    size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x < w)
        start_offset[x] = expert_id;
}

class MixtureOfExpertsTest : public ::testing::Test
{
protected:
    using DataType = float;

    static BufferManager::CudaStreamPtr mStream;
    static std::unique_ptr<BufferManager> mBufferManager;
    static int mDeviceCount;

    std::vector<BufferManager::IBufferPtr> managed_buffers;
    float* mInputProbabilities{};
    DataType* mInputTensor{};

    int mMaxSeqLen = 0;

    int mHiddenSize{};
    int mNumExperts{};
    int mK{};

    static void SetUpTestCase()
    {
        mDeviceCount = getDeviceCount();
        if (mDeviceCount > 0)
        {
            mStream = std::make_shared<CudaStream>();
            mBufferManager = std::make_unique<BufferManager>(mStream);
        }
        else
        {
            GTEST_SKIP();
        }
    }

    static void TearDownTestCase()
    {
        mBufferManager.reset();
        mStream.reset();
    }

    void SetUp() override
    {
        assert(mBufferManager);
        if (mDeviceCount == 0)
        {
            GTEST_SKIP();
        }
    }

    void TearDown()
    {
        managed_buffers.clear();
    }

    void initWeights(DataType* buffer, int w, int h, DataType scalar)
    {
        dim3 block(16, 16, 1);
        dim3 grid(divUp(w, block.x), divUp(h, block.y), mNumExperts);
        initWeightsKernel<DataType><<<grid, block, 0, mStream->get()>>>(buffer, w, h, scalar);
    }

    void initBias(DataType* buffer, int w)
    {
        dim3 block(256, 1, 1);
        dim3 grid(divUp(w, block.x), mNumExperts);
        initBiasToExpertIdKernel<DataType><<<grid, block, 0, mStream->get()>>>(buffer, w);
    }

    CutlassMoeFCRunner<DataType, DataType> mMoERunner{};
    char* mWorkspace{};
    DataType* mScaleProbs{};
    DataType* mExpertWeight1{};
    DataType* mExpertWeight2{};
    DataType* mExpertBias1{};
    DataType* mExpertBias2{};

    DataType* mTpExpertScratch{}; // Copy the experts here when slicing up inputs
    size_t mTpExpertScratchSize{};

    DataType* mExpertOutput{};
    DataType* mFinalOutput{};
    int* mSourceToExpandedMap;
    int* mSelectedExpert;
    bool* mFinished{};
    int mInterSize{};
    int mTotalTokens{};
    int mActiveRows{};

    bool mUseBias = true;

    tensorrt_llm::ActivationType mActType = tensorrt_llm::ActivationType::Relu;
    MOEExpertScaleNormalizationMode mNormMode = MOEExpertScaleNormalizationMode::NONE;

    int mExpertWDiag1 = 1;
    int mExpertWDiag2 = 2;

    template <class T>
    T* allocBuffer(size_t size)
    {
        managed_buffers.emplace_back(mBufferManager->gpu(size * sizeof(T)));
        T* ptr = static_cast<T*>(managed_buffers.back()->data());
        return ptr;
    }

    void initBuffersPermute(std::vector<std::vector<DataType>> h_hidden_states,
        std::vector<std::vector<float>> h_router_results, int hidden_size, int num_experts, int k,
        std::vector<uint8_t> finished, MOEParallelismConfig parallelism_config)
    {
        managed_buffers.clear();

        mHiddenSize = hidden_size;
        mInterSize = hidden_size * 4;
        mNumExperts = num_experts;
        mK = k;

        mTotalTokens = 0;
        std::vector<int> h_seq_lens;
        h_seq_lens.push_back(0);
        for (auto& sequence : h_hidden_states)
        {
            assert(sequence.size() % hidden_size == 0);
            int num_tokens = sequence.size() / hidden_size;
            h_seq_lens.emplace_back(h_seq_lens.back() + num_tokens);
            mTotalTokens += num_tokens;
            mMaxSeqLen = std::max(mMaxSeqLen, num_tokens);
        }

        size_t workspace_size = mMoERunner.getWorkspaceSize(
            mTotalTokens, mHiddenSize, mInterSize, mNumExperts, mK, mActType, parallelism_config);

        const auto stream = mStream->get();

        mWorkspace = allocBuffer<char>(workspace_size);
        check_cuda_error(hipMemsetAsync(mWorkspace, 0xD5, workspace_size, stream));
        const size_t expert_matrix_size = mNumExperts * mHiddenSize * mInterSize;

        mExpertWeight1 = allocBuffer<DataType>(expert_matrix_size);
        mExpertWeight2 = allocBuffer<DataType>(expert_matrix_size);

        mTpExpertScratchSize = 2 * expert_matrix_size / parallelism_config.tp_size;

        mExpertBias1 = nullptr;
        mExpertBias2 = nullptr;
        if (mUseBias)
        {
            // Allow space for the slice of bias1 in the scratch
            mTpExpertScratchSize += mNumExperts * mInterSize / parallelism_config.tp_size;
            mExpertBias1 = allocBuffer<DataType>(mNumExperts * mInterSize);
            mExpertBias2 = allocBuffer<DataType>(mNumExperts * mHiddenSize);

            check_cuda_error(hipMemsetAsync(mExpertBias1, 0x0, mNumExperts * mInterSize * sizeof(DataType), stream));
            check_cuda_error(hipMemsetAsync(mExpertBias2, 0x0, mNumExperts * mHiddenSize * sizeof(DataType), stream));
        }

        mExpertOutput = allocBuffer<DataType>(mTotalTokens * mHiddenSize * mK);

        mTpExpertScratch = nullptr;
        if (parallelism_config.tp_size > 1)
        {
            mTpExpertScratch = allocBuffer<DataType>(mTpExpertScratchSize);
        }

        mActiveRows = mTotalTokens;
        mFinished = nullptr;
        if (!finished.empty())
        {
            mFinished = allocBuffer<bool>(mTotalTokens);
            check_cuda_error(hipMemcpyAsync(
                mFinished, finished.data(), mTotalTokens * sizeof(bool), hipMemcpyHostToDevice, stream));
            static_assert(sizeof(bool) == sizeof(uint8_t), "Test assumes bool is interchangeable with uint8_t");
            mActiveRows = std::count(finished.begin(), finished.end(), 0);
        }

        mInputProbabilities = allocBuffer<float>(mTotalTokens * mNumExperts);
        mScaleProbs = allocBuffer<DataType>(mTotalTokens * mK);
        mInputTensor = allocBuffer<DataType>(mTotalTokens * mHiddenSize);
        mFinalOutput = allocBuffer<DataType>(mTotalTokens * mHiddenSize);

        mSourceToExpandedMap = allocBuffer<int>(mTotalTokens * mK);
        mSelectedExpert = allocBuffer<int>(mTotalTokens * mK);

        auto* input_probs_ptr = mInputProbabilities;
        for (auto& sequence : h_router_results)
        {
            check_cuda_error(hipMemcpyAsync(
                input_probs_ptr, sequence.data(), sequence.size() * sizeof(float), hipMemcpyHostToDevice, stream));
            input_probs_ptr += sequence.size();
        }

        auto* hidden_states_ptr = mInputTensor;
        for (auto& sequence : h_hidden_states)
        {
            check_cuda_error(hipMemcpyAsync(hidden_states_ptr, sequence.data(), sequence.size() * sizeof(DataType),
                hipMemcpyHostToDevice, stream));
            hidden_states_ptr += sequence.size();
        }

        // Init the diagonals of our matrix, this will set to the scalar value * expert_id
        initWeights(mExpertWeight1, mHiddenSize, mInterSize, mExpertWDiag1);
        initWeights(mExpertWeight2, mInterSize, mHiddenSize, mExpertWDiag2);

        if (mUseBias)
        {
            initBias(mExpertBias1, mInterSize);
            initBias(mExpertBias2, mHiddenSize);
        }

        check_cuda_error(hipStreamSynchronize(mStream->get()));
    }

    void resetOutBuffers()
    {
        auto stream = mStream->get();

        if (mTpExpertScratch)
            check_cuda_error(hipMemsetAsync(mTpExpertScratch, 0x0, mTpExpertScratchSize, stream));
        check_cuda_error(hipMemsetAsync(mFinalOutput, 0x0, mTotalTokens * mHiddenSize * sizeof(DataType), stream));
        check_cuda_error(hipMemsetAsync(mSourceToExpandedMap, 0x0, sizeof(int) * mTotalTokens * mK, stream));
        check_cuda_error(hipMemsetAsync(mSelectedExpert, 0x0, sizeof(int) * mTotalTokens * mK, stream));
        check_cuda_error(hipMemsetAsync(mScaleProbs, 0x0, sizeof(DataType) * mTotalTokens * mK, stream));
        check_cuda_error(
            hipMemsetAsync(mExpertOutput, 0x0, mTotalTokens * mHiddenSize * mK * sizeof(DataType), stream));

        check_cuda_error(hipStreamSynchronize(mStream->get()));
    }

    void runMoEPermute(std::vector<std::vector<DataType>> h_hidden_states,
        std::vector<std::vector<float>> h_router_results, int hidden_size, int num_experts, int k,
        std::vector<uint8_t> finished = {}, MOEParallelismConfig parallelism_config = {})
    {
        initBuffersPermute(std::move(h_hidden_states), std::move(h_router_results), hidden_size, num_experts, k,
            finished, parallelism_config);
        runMoEPermute(parallelism_config);
    }

    auto getWeights(MOEParallelismConfig parallelism_config)
    {
        if (parallelism_config.tp_size > 1)
        {
            const int tp_size = parallelism_config.tp_size;
            const int tp_rank = parallelism_config.tp_rank;

            const size_t matrix_size = mHiddenSize * mInterSize / tp_size;

            auto* weight_1 = mTpExpertScratch;
            auto* weight_2 = weight_1 + mNumExperts * matrix_size;
            auto* bias_1 = mUseBias ? weight_2 + mNumExperts * matrix_size : nullptr;

            // 2D memcpy just the slices we care about
            const size_t row_size_1 = matrix_size * sizeof(DataType);
            check_cuda_error(hipMemcpy2DAsync(weight_1, row_size_1, (uint8_t*) mExpertWeight1 + row_size_1 * tp_rank,
                row_size_1 * tp_size, row_size_1, mNumExperts, hipMemcpyDeviceToDevice, mStream->get()));

            const size_t row_size_2 = mInterSize / tp_size * sizeof(DataType);
            check_cuda_error(hipMemcpy2DAsync(weight_2, row_size_2, (uint8_t*) mExpertWeight2 + row_size_2 * tp_rank,
                row_size_2 * tp_size, row_size_2, mNumExperts * mHiddenSize, hipMemcpyDeviceToDevice, mStream->get()));

            if (mUseBias)
            {
                const size_t row_size_bias = mInterSize / tp_size * sizeof(DataType);
                check_cuda_error(
                    hipMemcpy2DAsync(bias_1, row_size_bias, (uint8_t*) mExpertBias1 + row_size_bias * tp_rank,
                        row_size_bias * tp_size, row_size_bias, mNumExperts, hipMemcpyDeviceToDevice, mStream->get()));
            }

            return std::tuple{weight_1, weight_2, bias_1, mExpertBias2};
        }
        else if (parallelism_config.ep_size > 1)
        {
            const size_t experts_per_node = mNumExperts / parallelism_config.ep_size;
            const size_t weight_matrix_size = mHiddenSize * mInterSize * experts_per_node;
            const size_t bias_fc1_size = mInterSize * experts_per_node;
            const size_t bias_fc2_size = mHiddenSize * experts_per_node;
            auto* weight1_ptr = mExpertWeight1 + weight_matrix_size * parallelism_config.ep_rank;
            auto* weight2_ptr = mExpertWeight2 + weight_matrix_size * parallelism_config.ep_rank;
            auto* bias1_ptr = mUseBias ? mExpertBias1 + bias_fc1_size * parallelism_config.ep_rank : nullptr;
            auto* bias2_ptr = mUseBias ? mExpertBias2 + bias_fc2_size * parallelism_config.ep_rank : nullptr;
            return std::tuple{weight1_ptr, weight2_ptr, bias1_ptr, bias2_ptr};
        }
        return std::tuple{mExpertWeight1, mExpertWeight2, mExpertBias1, mExpertBias2};
    }

    void runMoEPermute(MOEParallelismConfig parallelism_config)
    {
        // Clear the buffers to blank so we can assume zero if not written
        resetOutBuffers();

        const auto [weight1_ptr, weight2_ptr, bias1_ptr, bias2_ptr] = getWeights(parallelism_config);

        auto stream = mStream->get();
        mMoERunner.setTactic(std::nullopt);
        mMoERunner.runMoe(mInputTensor, mInputProbabilities, weight1_ptr, nullptr, bias1_ptr, mActType, weight2_ptr,
            nullptr, bias2_ptr, mTotalTokens, mHiddenSize, mInterSize / parallelism_config.tp_size, mNumExperts, mK,
            mWorkspace, mFinalOutput, mExpertOutput, mFinished, mActiveRows, mScaleProbs, mSourceToExpandedMap,
            mSelectedExpert, parallelism_config, mNormMode, stream);
        check_cuda_error(hipStreamSynchronize(mStream->get()));
    }

    template <class T>
    std::vector<T> getDataFromDevice(const T* in, size_t length)
    {
        std::vector<T> data(length);

        const auto stream = mStream->get();
        check_cuda_error(hipMemcpyAsync(data.data(), in, length * sizeof(T), hipMemcpyDeviceToHost, stream));
        check_cuda_error(hipStreamSynchronize(mStream->get()));

        return data;
    }

    auto maskSelectedExpertsForTP(const std::vector<int>& vector, int tp_size, int tp_rank)
    {
        std::vector<int> result;
        int num_experts_per_node = mNumExperts / tp_size;
        std::transform(vector.begin(), vector.end(), std::back_inserter(result),
            [=](int entry)
            {
                if (entry >= num_experts_per_node * tp_rank && entry < num_experts_per_node * (tp_rank + 1))
                    return entry;
                return mNumExperts;
            });
        return result;
    }

    void debugPrint()
    {
#define PRINT_CAST(array, size, cast)                                                                                  \
    do                                                                                                                 \
        if (array)                                                                                                     \
        {                                                                                                              \
            auto data = getDataFromDevice(array, size);                                                                \
            std::cout << #array << ": ";                                                                               \
            for (auto v : data)                                                                                        \
                std::cout << cast(v) << ", ";                                                                          \
            std::cout << std::endl;                                                                                    \
        }                                                                                                              \
    while (0)
#define PRINT(array, size) PRINT_CAST(array, size, )

        PRINT(mExpertWeight1, mNumExperts * mHiddenSize * mInterSize);
        PRINT(mExpertWeight2, mNumExperts * mHiddenSize * mInterSize);
        PRINT(mExpertBias1, mNumExperts * mInterSize);
        PRINT(mExpertBias2, mNumExperts * mHiddenSize);
        PRINT(mExpertOutput, mTotalTokens * mK * mHiddenSize);
        PRINT(mFinalOutput, mTotalTokens * mK * mHiddenSize);
        PRINT_CAST((uint8_t*) mFinished, mTotalTokens, (int) );
        PRINT(mInputProbabilities, mTotalTokens * mNumExperts);
        PRINT(mScaleProbs, mTotalTokens * mK);
        PRINT(mInputProbabilities, mTotalTokens * mNumExperts);
        PRINT(mInputTensor, mTotalTokens * mHiddenSize);
        PRINT(mSourceToExpandedMap, mTotalTokens * mK);
        PRINT(mSelectedExpert, mTotalTokens * mK);

#undef PRINT_CAST
#undef PRINT
    }

    DataType actfn(DataType in)
    {
        if (mActType == tensorrt_llm::ActivationType::Identity)
            return in;
        if (mActType == tensorrt_llm::ActivationType::Relu)
            return std::max(in, 0.0f);
        assert(false);
        return in;
    }

    DataType calcMLPVal(DataType input, int expert_id, bool final_bias = false)
    {
        if (expert_id >= mNumExperts)
            return 0;
        auto fc1 = input * mExpertWDiag1 + (DataType) (mUseBias ? expert_id : 0);
        auto activated = actfn(fc1) * mExpertWDiag2;
        return activated + (DataType) (final_bias ? expert_id : 0);
    }

    DataType calcMLPValWithFinalBias(DataType input, int expert_id)
    {
        return calcMLPVal(input, expert_id, mUseBias);
    }

    void comparePermuted(const std::vector<int>& expected_experts, const std::vector<int>& expected_permutation,
        const std::vector<DataType>& input_data)
    {
        auto states = getDataFromDevice(mExpertOutput, mTotalTokens * mK * mHiddenSize);

        // Loop for the number of times each token is duplicated
        for (int k_idx = 0; k_idx < mK; k_idx++)
        {
            for (int token_id = 0; token_id < mTotalTokens; token_id++)
            {
                // Permutation has the position of the first copy of all token,
                // followed by the position of the second copy of all tokens etc.
                const int permuted_position = expected_permutation[k_idx * mTotalTokens + token_id];

                // Expected experts has all the selected experts for token one,
                // followed by all the selected experts for token two etc.
                const int expert_id = expected_experts[token_id * mK + k_idx];

                // Compare the copied tokens with the projection applied
                for (int hidden_id = 0; hidden_id < mHiddenSize; hidden_id++)
                {
                    EXPECT_FLOAT_EQ(calcMLPVal(input_data[token_id * mHiddenSize + hidden_id], expert_id),
                        states[permuted_position * mHiddenSize + hidden_id])
                        << "Incorrect value at position: mK: " << k_idx << ", token: " << token_id
                        << ", permuted dest: " << permuted_position << ", expert id: " << expert_id;
                }
            }
        }
    }

    std::vector<DataType> softmax(const std::vector<DataType>& expected_probs)
    {
        std::vector<DataType> result;
        // All values we test are 0-1 so we can skip the normalization step
        std::transform(expected_probs.begin(), expected_probs.end(), std::back_inserter(result),
            [&](const DataType in)
            {
                auto res = exp(in);
                return res;
            });

        for (int token = 0; token < mTotalTokens; token++)
        {
            auto start = result.begin() + token * mNumExperts;
            auto end = start + mNumExperts;
            auto sum = std::accumulate(start, end, (DataType) 0);
            std::transform(start, end, start, [=](auto in) { return in / sum; });
        }

        return result;
    }

    void compareSoftmax(const std::vector<int>& expected_experts, const std::vector<DataType>& expected_probs,
        std::vector<DataType> scale_probs = {})
    {
        if (scale_probs.empty())
            scale_probs = getDataFromDevice(mScaleProbs, mTotalTokens * mK);
        auto softmax_probs = softmax(expected_probs);
        for (int token_id = 0; token_id < mTotalTokens; token_id++)
        {
            for (int k_idx = 0; k_idx < mK; k_idx++)
            {
                int selected_expert = expected_experts[token_id * mK + k_idx];
                if (selected_expert < mNumExperts) // Ignore 'finished' values
                {
                    ASSERT_FLOAT_EQ(
                        softmax_probs[token_id * mNumExperts + selected_expert], scale_probs[token_id * mK + k_idx])
                        << "Scales mismatched for token: " << token_id << " k: " << k_idx
                        << " selected_expert: " << selected_expert;
                }
            }
        }
    }

    void renormScales(DataType* probs, const int* experts)
    {
        if (mNormMode == MOEExpertScaleNormalizationMode::NONE)
            return;
        DataType sum = 0;
        for (int k_idx = 0; k_idx < mK; k_idx++)
        {
            sum += probs[experts[k_idx]];
        }
        DataType norm_factor = 1.0 / sum;
        for (int k_idx = 0; k_idx < mK; k_idx++)
        {
            probs[experts[k_idx]] *= norm_factor;
        }
    }

    void compareFinal(const std::vector<int>& expected_experts, const std::vector<DataType>& expected_probs,
        const std::vector<DataType>& input_data, std::vector<DataType> final_results = {})
    {
        if (final_results.empty())
            final_results = getDataFromDevice(mFinalOutput, mTotalTokens * mHiddenSize);

        auto softmax_probs = softmax(expected_probs);
        for (int token_id = 0; token_id < mTotalTokens; token_id++)
        {
            // Compare the copied tokens with the projection applied
            for (int hidden_id = 0; hidden_id < mHiddenSize; hidden_id++)
            {
                renormScales(&softmax_probs[token_id * mNumExperts], &expected_experts[token_id * mK]);

                DataType sum = 0.0f;
                // Loop for the number of times each token is duplicated
                for (int k_idx = 0; k_idx < mK; k_idx++)
                {
                    int selected_expert = expected_experts[token_id * mK + k_idx];
                    sum += calcMLPValWithFinalBias(input_data[token_id * mHiddenSize + hidden_id], selected_expert)
                        * softmax_probs[token_id * mNumExperts + selected_expert];
                }

                EXPECT_FLOAT_EQ(sum, final_results[token_id * mHiddenSize + hidden_id])
                    << "Incorrect final value at position: " << token_id * mHiddenSize + hidden_id;
            }
        }
    }

    void BasicPermuteTest(int k = 1);

    std::vector<int> calcPermuteMapExpertParallel(const std::vector<int>& expected_experts);
    void ExpertParallelTest(int k = 1);

    void TensorParallelTest(int k = 1);
};

BufferManager::CudaStreamPtr MixtureOfExpertsTest::mStream{};
std::unique_ptr<BufferManager> MixtureOfExpertsTest::mBufferManager{};
int MixtureOfExpertsTest::mDeviceCount{};

const int DEFAULT_HIDDEN_SIZE = 4;

void MixtureOfExpertsTest::BasicPermuteTest(int k)
{
    int hidden_size = DEFAULT_HIDDEN_SIZE;
    int num_experts = 4;
    int num_tokens = 3;

    std::vector<DataType> hidden_states(hidden_size * num_tokens, 0);
    std::iota(hidden_states.begin(), hidden_states.end(), 0.0f);

    std::vector<float> probs = {
        0.5, 0.1, 0.25, 0.15,   //
        0.03, 0.2, 0.07, 0.7,   //
        0.25, 0.21, 0.35, 0.19, //
    };

    runMoEPermute({hidden_states}, {probs}, hidden_size, num_experts, k);

    std::vector<int> expected_experts{0, 3, 2};
    if (k == 2)
        expected_experts = {0, 2, 3, 1, 2, 0};
    else if (k == 3)
        expected_experts = {0, 2, 3, 3, 1, 2, 2, 0, 1};

    auto selected_expert = getDataFromDevice(mSelectedExpert, num_tokens * k);
    EXPECT_EQ(selected_expert, expected_experts);

    auto proj_map = getDataFromDevice(mSourceToExpandedMap, num_tokens * k);
    // This is the final position of:
    // Token 1 Expert 1, T2E1, T3E1, T1E2, T2E2, T3E2
    std::vector<int> permute_map{0, 2, 1};
    if (k == 2)
        permute_map = {0, 5, 4, 3, 2, 1};
    if (k == 3)
        permute_map = {0, 8, 6, 4, 2, 1, 7, 5, 3};
    ASSERT_EQ(permute_map, proj_map);
    comparePermuted(selected_expert, permute_map, hidden_states);
    compareSoftmax(selected_expert, probs);
    compareFinal(selected_expert, probs, hidden_states);
}

TEST_F(MixtureOfExpertsTest, Permute)
{
    BasicPermuteTest();
}

TEST_F(MixtureOfExpertsTest, PermuteK2)
{
    BasicPermuteTest(2);
}

TEST_F(MixtureOfExpertsTest, PermuteK3)
{
    BasicPermuteTest(3);
}

TEST_F(MixtureOfExpertsTest, PermuteNoBias)
{
    mUseBias = false;
    BasicPermuteTest();
    BasicPermuteTest(2);
    BasicPermuteTest(3);
}

TEST_F(MixtureOfExpertsTest, PermuteRenormalization)
{
    mNormMode = tensorrt_llm::kernels::MOEExpertScaleNormalizationMode::RENORMALIZE;
    BasicPermuteTest();
    BasicPermuteTest(2);
    BasicPermuteTest(3);
}

TEST_F(MixtureOfExpertsTest, Finished)
{
    int hidden_size = DEFAULT_HIDDEN_SIZE;
    int num_experts = 4;
    int num_tokens = 3;
    int k = 2;

    std::vector<DataType> hidden_states(hidden_size * num_tokens, 0);
    std::iota(hidden_states.begin(), hidden_states.end(), 0.0f);

    std::vector<float> probs = {
        0.5, 0.1, 0.25, 0.15, //
        0.05, 0.2, 0.05, 0.7, //
        0.25, 0.2, 0.35, 0.2, //
    };

    runMoEPermute({hidden_states}, {probs}, hidden_size, num_experts, k, {0, 0, 1});

    auto selected_expert = getDataFromDevice(mSelectedExpert, num_tokens * k);
    // Token 1
    EXPECT_EQ(selected_expert[0], 0);
    EXPECT_EQ(selected_expert[1], 2);
    // Token 2
    EXPECT_EQ(selected_expert[2], 3);
    EXPECT_EQ(selected_expert[3], 1);
    // Token 3
    EXPECT_EQ(selected_expert[4], num_experts); // One past the end
    EXPECT_EQ(selected_expert[5], num_experts);

    auto proj_map = getDataFromDevice(mSourceToExpandedMap, num_tokens * k);
    // This is the final position of:
    // Token 1 Expert 1, T2E1, T3E1, T1E2, T2E2, T3E3
    std::vector<int> permute_map{0, 3, 4, 2, 1, 5};
    ASSERT_EQ(permute_map, proj_map);
    comparePermuted(selected_expert, permute_map, hidden_states);
    compareSoftmax(selected_expert, probs);
    compareFinal(selected_expert, probs, hidden_states);
}

std::vector<int> MixtureOfExpertsTest::calcPermuteMapExpertParallel(const std::vector<int>& expected_experts)
{
    std::vector<int> map(expected_experts.size());
    auto getInterleavedIndex = [this](int i) { return (i % mK) * mTotalTokens + i / mK; };
    int map_idx = 0;
    for (int expert = 0; expert <= mNumExperts; expert++)
    {
        for (int i = 0; i < map.size(); i++)
        {
            if (expected_experts[i] == expert)
                map[getInterleavedIndex(i)] = map_idx++;
        }
    }

    return map;
}

void MixtureOfExpertsTest::ExpertParallelTest(int k)
{
    int hidden_size = DEFAULT_HIDDEN_SIZE;
    int parallelism = 2;
    int num_experts = 4;
    int num_tokens = 3;

    std::vector<DataType> hidden_states(hidden_size * num_tokens, 0);
    std::iota(hidden_states.begin(), hidden_states.end(), 0.0f);

    std::vector<float> probs = {
        0.5, 0.1, 0.25, 0.15,   //
        0.03, 0.2, 0.07, 0.7,   //
        0.25, 0.21, 0.35, 0.19, //
    };

    std::vector<int> expected_experts{0, 3, 2};
    if (k == 2)
        expected_experts = {0, 2, 3, 1, 2, 0};
    else if (k == 3)
        expected_experts = {0, 2, 3, 3, 1, 2, 2, 0, 1};
    std::vector<DataType> results(hidden_states.size(), 0);
    for (int i = 0; i < parallelism; i++)
    {
        if (i == 0)
        {
            // Only need to init the inputs on the first iteration
            runMoEPermute({hidden_states}, {probs}, hidden_size, num_experts, k, {},
                MOEParallelismConfig::ExpertParallelism(parallelism, i));
        }
        else
        {
            runMoEPermute(MOEParallelismConfig::ExpertParallelism(parallelism, i));
        }

        auto selected_expert = getDataFromDevice(mSelectedExpert, num_tokens * k);
        // Experts should only be selected when we are on the right node
        // Note the index is [0,num_experts_per_node), so we offset the experts by the start for this node
        const int start_expert = i * (mNumExperts / parallelism);
        std::transform(selected_expert.begin(), selected_expert.end(), selected_expert.begin(),
            [&](int val) { return val == mNumExperts ? mNumExperts : val + start_expert; });
        auto masked_expected_experts = maskSelectedExpertsForTP(expected_experts, parallelism, i);
        ASSERT_EQ(selected_expert, masked_expected_experts);

        auto proj_map = getDataFromDevice(mSourceToExpandedMap, num_tokens * k);
        auto permute_map = calcPermuteMapExpertParallel(masked_expected_experts);
        ASSERT_EQ(permute_map, proj_map) << "Iteration " << i;
        comparePermuted(masked_expected_experts, permute_map, hidden_states);
        compareSoftmax(expected_experts, probs);

        // Do the final reduce
        auto iter_results = getDataFromDevice(mFinalOutput, num_tokens * hidden_size);
        std::transform(iter_results.cbegin(), iter_results.cend(), results.cbegin(), results.begin(), std::plus<>{});
    }

    compareFinal(expected_experts, probs, hidden_states, results);
}

TEST_F(MixtureOfExpertsTest, ExpertParallel)
{
    ExpertParallelTest();
}

TEST_F(MixtureOfExpertsTest, ExpertParallelK2)
{
    ExpertParallelTest(2);
}

TEST_F(MixtureOfExpertsTest, ExpertParallelNoBias)
{
    mUseBias = false;
    ExpertParallelTest();
    ExpertParallelTest(2);
}

TEST_F(MixtureOfExpertsTest, ExpertParallelRenorm)
{
    mNormMode = MOEExpertScaleNormalizationMode::RENORMALIZE;
    ExpertParallelTest();
    ExpertParallelTest(2);
}

void MixtureOfExpertsTest::TensorParallelTest(int k)
{
    int hidden_size = DEFAULT_HIDDEN_SIZE;
    int parallelism = 8;
    int num_experts = 4;
    int num_tokens = 3;

    std::vector<DataType> hidden_states(hidden_size * num_tokens, 0);
    std::iota(hidden_states.begin(), hidden_states.end(), 0.0f);

    std::vector<float> probs = {
        0.5, 0.1, 0.25, 0.15,   //
        0.03, 0.2, 0.07, 0.7,   //
        0.25, 0.21, 0.35, 0.19, //
    };

    std::vector<int> expected_experts{0, 3, 2};
    if (k == 2)
        expected_experts = {0, 2, 3, 1, 2, 0};
    else if (k == 3)
        expected_experts = {0, 2, 3, 3, 1, 2, 2, 0, 1};
    std::vector<DataType> results(hidden_states.size(), 0);
    for (int i = 0; i < parallelism; i++)
    {
        if (i == 0)
        {
            // Only need to init the inputs on the first iteration
            runMoEPermute({hidden_states}, {probs}, hidden_size, num_experts, k, {},
                MOEParallelismConfig::TensorParallelism(parallelism, i));
        }
        else
        {
            runMoEPermute(MOEParallelismConfig::TensorParallelism(parallelism, i));
        }

        auto selected_expert = getDataFromDevice(mSelectedExpert, num_tokens * k);
        EXPECT_EQ(selected_expert, expected_experts);

        auto proj_map = getDataFromDevice(mSourceToExpandedMap, num_tokens * k);
        std::vector<int> permute_map{0, 2, 1};
        if (k == 2)
            permute_map = {0, 5, 4, 3, 2, 1};
        if (k == 3)
            permute_map = {0, 8, 6, 4, 2, 1, 7, 5, 3};

        ASSERT_EQ(permute_map, proj_map) << "Iteration " << i;

        // Do the final reduce
        auto iter_results = getDataFromDevice(mFinalOutput, num_tokens * hidden_size);
        std::transform(iter_results.cbegin(), iter_results.cend(), results.cbegin(), results.begin(), std::plus<>{});
    }

    compareFinal(expected_experts, probs, hidden_states, results);
}

TEST_F(MixtureOfExpertsTest, TensorParallel)
{
    TensorParallelTest();
}

TEST_F(MixtureOfExpertsTest, TensorParallelK2)
{
    TensorParallelTest(2);
}

TEST_F(MixtureOfExpertsTest, TensorParallelK3)
{
    TensorParallelTest(3);
}

TEST_F(MixtureOfExpertsTest, TensorParallelNoBias)
{
    mUseBias = false;
    TensorParallelTest();
    TensorParallelTest(2);
    TensorParallelTest(3);
}

TEST_F(MixtureOfExpertsTest, TensorParallelRenorm)
{
    mNormMode = MOEExpertScaleNormalizationMode::RENORMALIZE;
    TensorParallelTest();
    TensorParallelTest(2);
    TensorParallelTest(3);
}
