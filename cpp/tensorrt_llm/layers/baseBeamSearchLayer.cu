#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/kernels/beamSearchPenaltyKernels.h"
#include "tensorrt_llm/layers/baseBeamSearchLayer.h"

using namespace tensorrt_llm::common;
using namespace tensorrt_llm::kernels;

namespace tensorrt_llm
{
namespace layers
{

__global__ void update_indir_cache_kernel(int* tgt_indir_cache, const int* src_indir_cache, const int** parent_ids,
    const bool* finished, const int* sequence_lengths, const int* input_lengths, int batch_dim, int local_batch_size,
    int beam_width, int max_seq_len)
{
    int time_step = threadIdx.x + blockIdx.x * blockDim.x;
    int bb_id = threadIdx.y + blockIdx.y * blockDim.y;
    const int current_step{sequence_lengths[bb_id] - 1}; // the sequence_lengths is updated, need to minus 1
    const int input_length{input_lengths == nullptr ? 0 : input_lengths[bb_id]};
    const int batch_id = bb_id / beam_width;
    const int beam_id = bb_id % beam_width;
    if (bb_id >= beam_width * local_batch_size || time_step < input_length || finished[bb_id])
    {
        return;
    }
    int time_step_circ = time_step % max_seq_len;
    // FIXME: we will remove all paddings later (@boyang)
    // Skip input paddings when updating the indir cache table.

    const int src_beam = parent_ids[batch_id][beam_id * max_seq_len + current_step];

    const uint32_t tgt_offset = batch_id * beam_width * max_seq_len + beam_id * max_seq_len + time_step_circ;
    const uint32_t src_offset = batch_id * beam_width * max_seq_len + src_beam * max_seq_len + time_step_circ;

    tgt_indir_cache[tgt_offset] = (time_step == current_step) ? beam_id : src_indir_cache[src_offset];
}

void update_indir_cache_kernelLauncher(int* tgt_indir_cache, const int* src_indir_cache, const int** parent_ids,
    const bool* finished, const int* sequence_lengths, const int* input_lengths, int batch_dim, int local_batch_size,
    int beam_width, int max_seq_len, hipStream_t stream)
{
    const dim3 block(32);
    // Update indirections steps [input_length[bb_id], sequence_lengths[bb_id]], included
    const dim3 grid((max_seq_len + block.x - 1) / block.x, local_batch_size * beam_width);
    update_indir_cache_kernel<<<grid, block, 0, stream>>>(tgt_indir_cache, src_indir_cache, parent_ids, finished,
        sequence_lengths, input_lengths, batch_dim, local_batch_size, beam_width, max_seq_len);
}

template <typename T>
BaseBeamSearchLayer<T>::BaseBeamSearchLayer(size_t vocab_size, size_t vocab_size_padded, hipStream_t stream,
    IAllocator* allocator, bool is_free_buffer_after_forward)
    : BaseLayer(stream, allocator, is_free_buffer_after_forward, nullptr)
    , vocab_size_(vocab_size)
    , vocab_size_padded_(vocab_size_padded)
{
}

template <typename T>
BaseBeamSearchLayer<T>::BaseBeamSearchLayer(BaseBeamSearchLayer<T> const& beam_search_layer)
    : BaseLayer(beam_search_layer)
    , vocab_size_(beam_search_layer.vocab_size_)
    , vocab_size_padded_(beam_search_layer.vocab_size_padded_)
    , topk_softmax_workspace_size_(beam_search_layer.topk_softmax_workspace_size_)
{
}

template <typename T>
BaseBeamSearchLayer<T>::~BaseBeamSearchLayer()
{
    TLLM_LOG_DEBUG(__PRETTY_FUNCTION__);
    freeBuffer();
}

template <typename T>
void BaseBeamSearchLayer<T>::freeBuffer()
{
    if (is_allocate_buffer_)
    {
        allocator_->free((void**) (&topk_softmax_workspace_));
        is_allocate_buffer_ = false;
    }
}

template <typename T>
void BaseBeamSearchLayer<T>::setupBase(SetupParams const& setupParams)
{
    mTemperature = (setupParams.temperature) ? setupParams.temperature->at(0) : 1.0f;
    mMinLength = (setupParams.min_length) ? setupParams.min_length->at(0) : 0;

    mRepetitionPenaltyType = RepetitionPenaltyType::None;
    mRepetitionPenalty = getDefaultPenaltyValue(mRepetitionPenaltyType);
    if (setupParams.repetition_penalty || setupParams.presence_penalty)
    {
        TLLM_CHECK_WITH_INFO(!(setupParams.repetition_penalty && setupParams.presence_penalty),
            "Found ambiguous parameters repetition_penalty and presence_penalty "
            "which are mutually exclusive. "
            "Please provide one of repetition_penalty or presence_penalty.");
        mRepetitionPenaltyType
            = setupParams.repetition_penalty ? RepetitionPenaltyType::Multiplicative : RepetitionPenaltyType::Additive;
        mRepetitionPenalty = mRepetitionPenaltyType == RepetitionPenaltyType::Multiplicative
            ? setupParams.repetition_penalty->at(0)
            : setupParams.presence_penalty->at(0);
    }
}

template <typename T>
void BaseBeamSearchLayer<T>::forward(BeamSearchOutputParams& outputs, ForwardParams const& params)
{
    TLLM_LOG_DEBUG("%s", __PRETTY_FUNCTION__);
    Tensor& output_ids_ptr = outputs.output_ids_ptr;

    const auto batch_size = static_cast<std::int32_t>(output_ids_ptr.shape[0]);
    const auto beam_width = static_cast<std::int32_t>(output_ids_ptr.shape[1]);
    const auto max_seq_len = static_cast<std::int32_t>(output_ids_ptr.shape[2]);
    allocateBuffer(batch_size, beam_width);

    TLLM_CHECK_WITH_INFO(params.ite == 0, "Pipeline Parallelism is not supported yet !");

    const int ite{params.ite};
    Tensor const& logits = params.logits;
    const auto local_batch_size = logits.shape[0];

    const T* embedding_bias = params.embedding_bias ? params.embedding_bias->template getPtr<const T>() : nullptr;

    auto* end_ids = params.end_ids.template getPtr<const int>();
    auto* const input_lengths = params.input_lengths ? params.input_lengths->template getPtr<const int>() : nullptr;
    int* sequence_length = (outputs.sequence_length) ? outputs.sequence_length->template getPtr<int>() : nullptr;

    invokeAddBiasApplyPenalties(logits.getPtr<T>(), output_ids_ptr.template getPtr<const int*>(),
        outputs.parent_ids_ptr.template getPtr<const int*>(), input_lengths, sequence_length, embedding_bias, ite,
        local_batch_size, batch_size, beam_width, vocab_size_, vocab_size_padded_, end_ids, mTemperature,
        mRepetitionPenalty, mRepetitionPenaltyType, mMinLength, max_seq_len, stream_);
    sync_check_cuda_error();

    invokeSoftMax(outputs, params);

    if (beam_width > 1)
    {
        update_indir_cache_kernelLauncher(outputs.tgt_cache_indirection.template getPtr<int>(),
            params.src_cache_indirection.template getPtr<const int>(),
            outputs.parent_ids_ptr.template getPtr<const int*>(), outputs.finished->template getPtr<const bool>(),
            sequence_length, input_lengths, batch_size, local_batch_size, beam_width, max_seq_len, stream_);
        sync_check_cuda_error();
    }
    sync_check_cuda_error();
    if (is_free_buffer_after_forward_)
    {
        freeBuffer();
    }
    sync_check_cuda_error();
}

template class BaseBeamSearchLayer<float>;
template class BaseBeamSearchLayer<half>;

} // namespace layers
} // namespace tensorrt_llm
