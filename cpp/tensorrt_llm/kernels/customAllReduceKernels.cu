#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "customAllReduceKernels.h"
#include "tensorrt_llm/common/cudaTypeUtils.cuh"
#include <tuple>

namespace tensorrt_llm::kernels
{

using tensorrt_llm::common::hadd2;
using tensorrt_llm::common::datatype_enum;
using tensorrt_llm::common::divUp;

////////////////////////////////////////////////////////////////////////////////////////////////////

static inline __device__ uint32_t myHadd2(const uint32_t& a, const uint32_t& b)
{
    uint32_t c;
    asm volatile("add.f16x2 %0, %1, %2;\n" : "=r"(c) : "r"(a), "r"(b));
    return c;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

static inline __device__ uint32_t fadd(const uint32_t& a, const uint32_t& b)
{
    uint32_t c;
    asm volatile("add.f32 %0, %1, %2;\n" : "=r"(c) : "r"(a), "r"(b));
    return c;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

static inline __device__ void st_flag_release(uint32_t& flag, uint32_t* flag_addr)
{
#if __CUDA_ARCH__ >= 700
    asm volatile("st.global.release.sys.b32 [%1], %0;" ::"r"(flag), "l"(flag_addr));
#else
    __threadfence_system();
    asm volatile("st.global.volatile.b32 [%1], %0;" ::"r"(flag), "l"(flag_addr));
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////

static inline __device__ void ld_flag_acquire(uint32_t& flag, uint32_t* flag_addr)
{
#if __CUDA_ARCH__ >= 700
    asm volatile("ld.global.acquire.sys.b32 %0, [%1];" : "=r"(flag) : "l"(flag_addr));
#else
    asm volatile("ld.global.volatile.b32 %0, [%1];" : "=r"(flag) : "l"(flag_addr));
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////

// Type Converter that packs data format to 128 bits data type
template <typename T>
struct ARTypeConverter
{
    using Type = uint4;
};

#ifdef ENABLE_BF16
template <>
struct ARTypeConverter<__hip_bfloat16>
{
    using Type = bf168;
};
#endif

// add two 128b data
template <typename T_IN, typename T_COMP>
inline __device__ T_IN add128b(T_IN a, T_IN b);

template <>
inline __device__ uint4 add128b<uint4, uint16_t>(uint4 a, uint4 b)
{
    uint4 c;
    c.x = myHadd2(a.x, b.x);
    c.y = myHadd2(a.y, b.y);
    c.z = myHadd2(a.z, b.z);
    c.w = myHadd2(a.w, b.w);
    return c;
}

template <>
inline __device__ uint4 add128b<uint4, uint32_t>(uint4 a, uint4 b)
{
    uint4 c;
    c.x = fadd(a.x, b.x);
    c.y = fadd(a.y, b.y);
    c.z = fadd(a.z, b.z);
    c.w = fadd(a.w, b.w);
    return c;
}

#ifdef ENABLE_BF16
template <>
inline __device__ bf168 add128b<bf168, __hip_bfloat16>(bf168 a, bf168 b)
{
    bf168 c;
    c.x = hadd2(a.x, b.x);
    c.y = hadd2(a.y, b.y);
    c.z = hadd2(a.z, b.z);
    c.w = hadd2(a.w, b.w);
    return c;
}
#endif

// init 128bits data with 0
template <typename T>
inline __device__ T init_packed_type();

template <>
inline __device__ uint4 init_packed_type()
{
    return make_uint4(0u, 0u, 0u, 0u);
}

#ifdef ENABLE_BF16
template <>
inline __device__ bf168 init_packed_type()
{
    bf168 val;
    uint4& val_u = reinterpret_cast<uint4&>(val);
    val_u = make_uint4(0u, 0u, 0u, 0u);
    return val;
}
#endif

__inline__ __device__ void multi_gpu_barrier(
    uint32_t** signals, const uint32_t flag, const size_t rank, const size_t world_size, const int tidx, const int bidx)
{
    // At the end of the function, we now that has least block 0 from all others GPUs have reached that point.
    volatile uint32_t* my_signals = signals[rank];
    if (tidx < world_size)
    {
        // The 1st block notifies the other ranks.
        if (bidx == 0)
        {
            signals[tidx][rank] = flag;
        }

        // Busy-wait until all ranks are ready.
        while (my_signals[tidx] != flag)
        {
        }
    }

    // Make sure we can move on...
    __syncthreads();
}

__global__ void multiGpuBarrierKernel(AllReduceParams params)
{
    multi_gpu_barrier(params.peer_barrier_ptrs_out, params.barrier_flag, params.local_rank, params.ranks_per_node,
        threadIdx.x, blockIdx.x);
}

template <typename T, int RANKS_PER_NODE>
static __global__ void oneShotAllReduceKernel(AllReduceParams params)
{
    const int bidx = blockIdx.x;
    const int tidx = threadIdx.x;

    // The number of elements packed into one for comms
    static constexpr int NUM_ELTS = std::is_same<T, uint32_t>::value ? 4 : 8;

    // Packed data type for comms
    using PackedType = typename ARTypeConverter<T>::Type;

    multi_gpu_barrier(params.peer_barrier_ptrs_in, params.barrier_flag, params.local_rank, RANKS_PER_NODE, tidx, bidx);

    // The source pointers. Distributed round-robin for the different warps.
    const T* src_d[RANKS_PER_NODE];
#pragma unroll
    for (int ii = 0; ii < RANKS_PER_NODE; ++ii)
    {
        int rank = (params.local_rank + ii) % RANKS_PER_NODE;
        src_d[ii] = reinterpret_cast<T*>(params.peer_comm_buffer_ptrs[rank]);
    }

    // The location in the destination array (load 8 fp16 or load 4 fp32 using LDG.128).
    size_t offset = bidx * params.elts_per_block + tidx * NUM_ELTS;
    // The end of the segment computed by that block.
    size_t max_offset = std::min((bidx + 1) * params.elts_per_block, params.elts_per_rank);

    // Each block accumulates the values from the different GPUs on the same node.
    for (size_t iter_offset = offset; iter_offset < max_offset; iter_offset += blockDim.x * NUM_ELTS)
    {
        // Iterate over the different ranks/devices on the node to load the values.
        PackedType vals[RANKS_PER_NODE];
#pragma unroll
        for (int ii = 0; ii < RANKS_PER_NODE; ++ii)
        {
            vals[ii] = reinterpret_cast<const PackedType*>(&src_d[ii][iter_offset])[0];
        }

        // Sum the values from the different ranks.
        PackedType sums = init_packed_type<PackedType>();
#pragma unroll
        for (int ii = 0; ii < RANKS_PER_NODE; ++ii)
        {
            sums = add128b<PackedType, T>(sums, vals[ii]);
        }

        // Store to the destination buffer.
        reinterpret_cast<PackedType*>(&reinterpret_cast<T*>(params.local_output_buffer_ptr)[iter_offset])[0] = sums;
    }
}

template <typename T, int RANKS_PER_NODE>
static __global__ void twoShotAllReduceKernel(AllReduceParams params)
{

    // The block index.
    const int bidx = blockIdx.x;
    // The thread index with the block.
    const int tidx = threadIdx.x;

    // The number of elements packed into one for comms
    static constexpr int NUM_ELTS = std::is_same<T, uint32_t>::value ? 4 : 8;

    // Packed data type for comms
    using PackedType = typename ARTypeConverter<T>::Type;

    // The location in the destination array (load 8 fp16 or load 4 fp32 using LDG.128).
    const size_t block_offset = bidx * params.elts_per_block + tidx * NUM_ELTS;
    const size_t block_start = params.rank_offset + block_offset;
    // The end of the segment computed by that block.
    size_t max_offset = min(block_start + params.elts_per_block, params.rank_offset + params.elts_per_rank);

    multi_gpu_barrier(params.peer_barrier_ptrs_in, params.barrier_flag, params.local_rank, RANKS_PER_NODE, tidx, bidx);

    // The source pointers. Distributed round-robin for the different warps.
    T* src_d[RANKS_PER_NODE];
    // The destination ranks for round-robin gathering
    size_t dst_rank[RANKS_PER_NODE];
#pragma unroll
    for (int ii = 0; ii < RANKS_PER_NODE; ++ii)
    {
        int rank = (params.local_rank + ii) % RANKS_PER_NODE;
        src_d[ii] = reinterpret_cast<T*>(params.peer_comm_buffer_ptrs[rank]);
        dst_rank[ii] = rank;
    }

    // Each block accumulates the values from the different GPUs on the same node.
    for (size_t local_offset = block_start; local_offset < max_offset; local_offset += blockDim.x * NUM_ELTS)
    {

        // Iterate over the different ranks/devices on the node to load the values.
        PackedType vals[RANKS_PER_NODE];
#pragma unroll
        for (int ii = 0; ii < RANKS_PER_NODE; ++ii)
        {
            vals[ii] = reinterpret_cast<const PackedType*>(&src_d[ii][local_offset])[0];
        }

        // Sum the values from the different ranks.
        PackedType sums = init_packed_type<PackedType>();
#pragma unroll
        for (int ii = 0; ii < RANKS_PER_NODE; ++ii)
        {
            sums = add128b<PackedType, T>(sums, vals[ii]);
        }

        // Store to the local buffer.
        reinterpret_cast<PackedType*>(&src_d[0][local_offset])[0] = sums;
    }

    // sync threads to make sure all block threads have the sums
    __syncthreads();

    // barriers among the blocks with the same idx (release-acquire semantics)
    if (tidx < RANKS_PER_NODE)
    {
        // The all blocks notifies the other ranks.
        uint32_t flag_block_offset = RANKS_PER_NODE + bidx * RANKS_PER_NODE;
        st_flag_release(params.barrier_flag, params.peer_barrier_ptrs_in[tidx] + flag_block_offset + params.local_rank);

        // Busy-wait until all ranks are ready.
        uint32_t rank_barrier = 0;
        uint32_t* peer_barrier_d = params.peer_barrier_ptrs_in[params.local_rank] + flag_block_offset + tidx;
        do
        {
            ld_flag_acquire(rank_barrier, peer_barrier_d);
        } while (rank_barrier != params.barrier_flag);
    }

    // sync threads to make sure all other ranks has the final partial results
    __syncthreads();

    size_t max_block_offset = min(block_offset + params.elts_per_block, params.elts_per_rank);
    // Gather all needed elts from other intra-node ranks
    for (size_t local_offset = block_offset; local_offset < max_block_offset; local_offset += blockDim.x * NUM_ELTS)
    {
#pragma unroll
        for (int ii = 0; ii < RANKS_PER_NODE; ++ii)
        {
            // use round-robin gathering from other ranks
            size_t offset_rank = dst_rank[ii] * params.elts_per_rank + local_offset;
            if (offset_rank >= params.elts_total)
            {
                continue;
            }
            reinterpret_cast<PackedType*>(&reinterpret_cast<T*>(params.local_output_buffer_ptr)[offset_rank])[0]
                = reinterpret_cast<PackedType*>(&src_d[ii][offset_rank])[0];
        }
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

std::tuple<int, int> kernelLaunchConfig(AllReduceStrategyType algo, AllReduceParams& param, size_t elts_per_thread)
{
    TLLM_CHECK(param.elts_total % elts_per_thread == 0);

    int blocks_per_grid = 1, threads_per_block = DEFAULT_BLOCK_SIZE;

    const size_t total_threads = param.elts_total / elts_per_thread;
    switch (algo)
    {
    case AllReduceStrategyType::ONESHOT:
    {     // one stage all reduce algo
        if (total_threads <= DEFAULT_BLOCK_SIZE)
        { // local reduce
            threads_per_block = WARP_SIZE * divUp(total_threads, WARP_SIZE);
            blocks_per_grid = 1;
        }
        else
        { // local reduce
            threads_per_block = DEFAULT_BLOCK_SIZE;
            blocks_per_grid = divUp(total_threads, DEFAULT_BLOCK_SIZE);
            blocks_per_grid = std::min(static_cast<int>(MAX_ALL_REDUCE_BLOCKS), blocks_per_grid);
        }
        param.elts_per_rank = param.elts_total;
        param.elts_per_block = elts_per_thread * divUp(param.elts_per_rank, elts_per_thread * blocks_per_grid);
        break;
    }
    case AllReduceStrategyType::TWOSHOT:
    { // two stage all reduce algo
        const size_t elts_per_rank = param.elts_total / param.ranks_per_node;
        TLLM_CHECK(elts_per_rank % elts_per_thread == 0);

        size_t total_threads = elts_per_rank / elts_per_thread;
        total_threads = WARP_SIZE * ((total_threads + WARP_SIZE - 1) / WARP_SIZE);
        TLLM_CHECK(total_threads % WARP_SIZE == 0);

        while (total_threads % blocks_per_grid != 0 || total_threads / blocks_per_grid > DEFAULT_BLOCK_SIZE)
        {
            blocks_per_grid += 1;
        }

        threads_per_block = total_threads / blocks_per_grid;

        // NOTE: need to adjust here
        if (blocks_per_grid > MAX_ALL_REDUCE_BLOCKS)
        {
            size_t iter_factor = 1;
            while (blocks_per_grid / iter_factor > MAX_ALL_REDUCE_BLOCKS || blocks_per_grid % iter_factor)
            {
                iter_factor += 1;
            }
            blocks_per_grid /= iter_factor;
        }
        param.elts_per_rank = param.elts_total / param.ranks_per_node;
        param.elts_per_block = param.elts_per_rank / blocks_per_grid;
        param.elts_per_block = elts_per_thread * divUp(param.elts_per_block, elts_per_thread);
        param.rank_offset = param.rank * param.elts_per_rank;
        break;
    }
    default: TLLM_THROW("Algorithm not supported here.");
    }

    return std::make_tuple(blocks_per_grid, threads_per_block);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename T, int RANKS_PER_NODE>
void dispatchARKernels(
    AllReduceStrategyType algo, AllReduceParams& param, int blocks_per_grid, int threads_per_block, hipStream_t stream)
{
    if (algo == AllReduceStrategyType::ONESHOT)
    {
        oneShotAllReduceKernel<T, RANKS_PER_NODE><<<blocks_per_grid, threads_per_block, 0, stream>>>(param);
    }
    else
    {
        twoShotAllReduceKernel<T, RANKS_PER_NODE><<<blocks_per_grid, threads_per_block, 0, stream>>>(param);
    }
}

template <typename T>
void invokeOneOrTwoShotAllReduceKernel(AllReduceParams& param, AllReduceStrategyType strat, hipStream_t stream)
{
    TLLM_CHECK(strat == AllReduceStrategyType::ONESHOT || strat == AllReduceStrategyType::TWOSHOT);
    sync_check_cuda_error();

    size_t elts_per_thread = 16 / sizeof(T);
    auto [blocks_per_grid, threads_per_block] = kernelLaunchConfig(strat, param, elts_per_thread);
    switch (param.ranks_per_node)
    {
    case 2: dispatchARKernels<T, 2>(strat, param, blocks_per_grid, threads_per_block, stream); break;
    case 4: dispatchARKernels<T, 4>(strat, param, blocks_per_grid, threads_per_block, stream); break;
    case 6: dispatchARKernels<T, 6>(strat, param, blocks_per_grid, threads_per_block, stream); break;
    case 8: dispatchARKernels<T, 8>(strat, param, blocks_per_grid, threads_per_block, stream); break;
    default: break;
    }
    sync_check_cuda_error();
}

void invokeMultiGpuBarrier(AllReduceParams& param, hipStream_t stream)
{
    multiGpuBarrierKernel<<<1, param.ranks_per_node, 0, stream>>>(param);
}

AllReduceParams AllReduceParams::deserialize(const int32_t* buffer, size_t tpSize, size_t tpRank, uint32_t flag_value)
{
    void* const* buffer_ptrs = reinterpret_cast<void* const*>(buffer);
    AllReduceParams params;

    for (int i = 0; i < tpSize; ++i)
    {
        params.peer_comm_buffer_ptrs[i] = buffer_ptrs[i];
    }
    for (int i = 0; i < tpSize; ++i)
    {
        params.peer_barrier_ptrs_in[i] = reinterpret_cast<uint32_t*>(buffer_ptrs[tpSize + i]);
    }
    for (int i = 0; i < tpSize; ++i)
    {
        params.peer_barrier_ptrs_out[i] = reinterpret_cast<uint32_t*>(buffer_ptrs[2 * tpSize + i]);
    }
    params.barrier_flag = flag_value;
    params.ranks_per_node = tpSize;
    params.rank = tpRank;
    params.local_rank = tpRank;

    return params;
}

void customAllReduce(kernels::AllReduceParams& params, void* data, size_t elts, size_t size_per_elem,
    datatype_enum dataType, AllReduceStrategyType strat, hipStream_t stream)
{
    params.local_output_buffer_ptr = data;
    params.elts_total = elts;

    if (dataType == datatype_enum::TYPE_FP32)
    {
        using T = CustomARCommTypeConverter<float>::Type;
        kernels::invokeOneOrTwoShotAllReduceKernel<T>(params, strat, stream);
    }
    else if (dataType == datatype_enum::TYPE_FP16)
    {
        using T = CustomARCommTypeConverter<half>::Type;
        kernels::invokeOneOrTwoShotAllReduceKernel<T>(params, strat, stream);
    }
    else if (dataType == datatype_enum::TYPE_BF16)
    {
        using T = CustomARCommTypeConverter<__hip_bfloat16>::Type;
        kernels::invokeOneOrTwoShotAllReduceKernel<T>(params, strat, stream);
    }
    else
    {
        TLLM_THROW("Unsupported dataType for customAllReduce");
    }
}

} // namespace tensorrt_llm::kernels
