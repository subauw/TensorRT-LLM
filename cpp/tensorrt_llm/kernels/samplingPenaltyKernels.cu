#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <assert.h>
#include <float.h>

#include "tensorrt_llm/common/assert.h"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/kernels/samplingPenaltyKernels.h"

namespace tensorrt_llm
{
namespace kernels
{

// TODO Add half2 implementation
template <typename T>
__global__ void applyTemperaturePenalty(T* logits, const T* bias, const float temperatureInverse, const int m,
    const int vocabSize, const int vocabSizePadded)
{
    const bool IS_FP16 = std::is_same<T, half>::value;
    const T MAX_T_VAL = (IS_FP16) ? 65504.F : FLT_MAX;
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < m * vocabSizePadded;
         index += blockDim.x * gridDim.x)
    {
        T biasVal = bias == nullptr ? (T) (0.0f) : bias[index % vocabSizePadded];
        if (index % vocabSizePadded < vocabSize)
        {
            logits[index] = (logits[index] + biasVal) * (T) temperatureInverse;
        }
        else
        {
            logits[index] = -MAX_T_VAL;
        }
    }
}

template <>
__global__ void applyTemperaturePenalty(half2* logits, const half2* bias, const float temperatureInverse,
    const int batchSize, const int vocabSize, const int vocabSizePaddeded)
{
    assert(vocabSize % 2 == 0);
    assert(vocabSizePaddeded % 2 == 0);
    const half2 maskVal = __float2half2_rn(-65504.0f);
    const half2 tempInv = __float2half2_rn(temperatureInverse);

    const int halfVocabSize = vocabSize / 2;
    const int halfVocabSizePaddeded = vocabSizePaddeded / 2;
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < batchSize * halfVocabSizePaddeded;
         index += blockDim.x * gridDim.x)
    {
        int vocabIdx = index % halfVocabSizePaddeded;
        half2 logit = vocabIdx < halfVocabSize ? __ldg(&logits[index]) : maskVal;
        if (vocabIdx < halfVocabSize)
        {
            if (bias != nullptr)
            {
                logit = __hadd2(logit, bias[vocabIdx]);
            }
            logits[index] = __hmul2(logit, tempInv);
        }
    }
}

template <typename T>
void invokeApplyTemperaturePenalty(T* logits, const T* bias, const float temperature, const int batchSize,
    const int vocabSize, const int vocabSizePadded, hipStream_t stream)
{
    dim3 block(min(vocabSizePadded, 1024));
    dim3 grid(min(batchSize * vocabSizePadded / block.x, 65536));
    const T temperatureInverse = (T) (1.f / (temperature + 1e-6f));
    if (std::is_same<T, half>::value && vocabSize % 2 == 0 && vocabSizePadded % 2 == 0)
    {
        applyTemperaturePenalty<<<grid, block, 0, stream>>>(reinterpret_cast<half2*>(logits),
            reinterpret_cast<const half2*>(bias), temperatureInverse, batchSize, vocabSize, vocabSizePadded);
    }
    else
    {
        applyTemperaturePenalty<T>
            <<<grid, block, 0, stream>>>(logits, bias, temperatureInverse, batchSize, vocabSize, vocabSizePadded);
    }
}

template void invokeApplyTemperaturePenalty(float* logits, const float* bias, const float temperature,
    const int batchSize, const int vocabSize, const int vocabSizePadded, hipStream_t stream);

template void invokeApplyTemperaturePenalty(half* logits, const half* bias, const float temperature,
    const int batchSize, const int vocabSize, const int vocabSizePadded, hipStream_t stream);

template <typename T>
__global__ void batchApplyTemperaturePenalty(T* logits, const T* bias, const float* temperatures, const int batchSize,
    const int vocabSize, const int vocabSizePadded)
{
    // TODO: Add macro or device function to get MAX_T_VAL.
    const bool IS_FP16 = std::is_same<T, half>::value;
    const T MAX_T_VAL = (IS_FP16) ? 65504.F : FLT_MAX;
    extern __shared__ float invTemperatures[];
    if (threadIdx.x < batchSize)
    {
        invTemperatures[threadIdx.x] = 1.0f / (temperatures[threadIdx.x] + 1e-6f);
    }
    __syncthreads();

    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < batchSize * vocabSizePadded;
         index += blockDim.x * gridDim.x)
    {
        int batchIdx = index / vocabSizePadded;
        int vocabIdx = index % vocabSizePadded;
        T logit = (vocabIdx < vocabSize) ? logits[index] : -MAX_T_VAL;
        if (vocabIdx < vocabSize)
        {
            if (bias != nullptr)
            {
                logit += bias[vocabIdx];
            }
            logit *= invTemperatures[batchIdx];
        }
        logits[index] = logit;
    }
}

__global__ void batchApplyTemperaturePenalty_h2(half2* logits, const half2* bias, const float* temperatures,
    const int batchSize, const int vocabSize, const int vocabSizePaddeded)
{
    assert(vocabSize % 2 == 0);
    assert(vocabSizePaddeded % 2 == 0);
    extern __shared__ half2 h2InvTemperatures[];
    if (threadIdx.x < batchSize)
    {
        h2InvTemperatures[threadIdx.x] = __float2half2_rn(1.f / (temperatures[threadIdx.x] + 1e-6f));
    }
    __syncthreads();

    const half2 maskVal = __float2half2_rn(-65504.0f);
    const int halfVocabSize = vocabSize / 2;
    const int halfVocabSizePaddeded = vocabSizePaddeded / 2;
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < batchSize * halfVocabSizePaddeded;
         index += blockDim.x * gridDim.x)
    {
        int batchIdx = index / halfVocabSizePaddeded;
        int vocabIdx = index % halfVocabSizePaddeded;
        half2 logit = vocabIdx < halfVocabSize ? __ldg(&logits[index]) : maskVal;
        if (vocabIdx < halfVocabSize)
        {
            if (bias != nullptr)
            {
                logit = __hadd2(logit, bias[vocabIdx]);
            }
            logits[index] = __hmul2(logit, h2InvTemperatures[batchIdx]);
        }
    }
}

template <typename T>
void invokeBatchApplyTemperaturePenalty(T* logits, const T* bias, const float* temperatures, const int batchSize,
    const int vocabSize, const int vocabSizePadded, hipStream_t stream)
{
    TLLM_LOG_DEBUG("%s start", __PRETTY_FUNCTION__);
    dim3 block(min(vocabSizePadded, 1024));
    dim3 grid(min(batchSize * vocabSizePadded / block.x, 65536));
    if (std::is_same<T, half>::value && vocabSize % 2 == 0 && vocabSizePadded % 2 == 0)
    {
        size_t smemSize = sizeof(half2) * batchSize;
        batchApplyTemperaturePenalty_h2<<<grid, block, smemSize, stream>>>(reinterpret_cast<half2*>(logits),
            reinterpret_cast<const half2*>(bias), temperatures, batchSize, vocabSize, vocabSizePadded);
    }
    else
    {
        size_t smemSize = sizeof(float) * batchSize;
        batchApplyTemperaturePenalty<T>
            <<<grid, block, smemSize, stream>>>(logits, bias, temperatures, batchSize, vocabSize, vocabSizePadded);
    }
}

template void invokeBatchApplyTemperaturePenalty(float* logits, const float* bias, const float* temperatures,
    const int batchSize, const int vocabSize, const int vocabSizePadded, hipStream_t stream);

template void invokeBatchApplyTemperaturePenalty(half* logits, const half* bias, const float* temperatures,
    const int batchSize, const int vocabSize, const int vocabSizePadded, hipStream_t stream);

template <typename T>
__global__ void batchApplyRepetitionPenalty(T* logits, const float* repetition_penalties,
    const float* presence_penalties, const float* frequency_penalties, const bool use_repetition,
    const bool use_presence, const bool use_frequency, const int** outputIds, const int* sequenceLengths,
    const int batchSize, const int vocabSize, const int maxSeqLen)
{
    extern __shared__ int penaltyIndices[];
    float* penaltyLogits;
    float repetition_penalty, presence_penalty, neg_frequency_penalty;
    if (use_repetition || use_presence)
    {
        penaltyLogits = (float*) (penaltyIndices + maxSeqLen);
    }
    const int batchIdx = blockIdx.x;
    if (use_repetition)
    {
        repetition_penalty = repetition_penalties[batchIdx];
    }
    if (use_presence)
    {
        presence_penalty = presence_penalties[batchIdx];
    }
    if (use_frequency)
    {
        neg_frequency_penalty = -frequency_penalties[batchIdx];
    }
    const int currentStep = sequenceLengths[batchIdx];

    logits += batchIdx * vocabSize;

    // Phase 1. Find indices to penalize and keep the penalized values of the combination of repetition and presence
    // penalty.
    for (int index = threadIdx.x; index < currentStep; index += blockDim.x)
    {
        // outputIds shape: (batchSize, input_len + output_len)
        int penaltyIndex = outputIds[batchIdx][blockIdx.y * maxSeqLen + index];
        penaltyIndices[index] = penaltyIndex;
        if (penaltyIndex >= vocabSize)
        {
            continue;
        }
        if (use_repetition || use_presence)
        {
            float logit = (float) logits[penaltyIndex];
            if (use_repetition)
            {
                logit = logit < 0.0f ? logit * repetition_penalty : logit / repetition_penalty;
            }
            if (use_presence)
            {
                logit -= presence_penalty;
            }
            penaltyLogits[index] = logit;
        }
    }

    // Phase 2. Apply the combination of repetition and presence penalty.
    if (use_repetition || use_presence)
    {
        if (blockDim.x > 32)
        {
            __syncthreads();
        }

        for (int index = threadIdx.x; index < currentStep; index += blockDim.x)
        {
            if (penaltyIndices[index] >= vocabSize)
            {
                continue;
            }
            logits[penaltyIndices[index]] = penaltyLogits[index];
        }
    }

    // Phase 3. Apply the frequency penalty.
    if (use_frequency)
    {
        if (blockDim.x > 32)
        {
            __syncthreads();
        }

        for (int index = threadIdx.x; index < currentStep; index += blockDim.x)
        {
            if (penaltyIndices[index] >= vocabSize)
            {
                continue;
            }
            atomicAdd(&logits[penaltyIndices[index]], neg_frequency_penalty);
        }
    }
}

template <typename T>
void invokeBatchApplyRepetitionPenalty(T* logits, const float* repetition_penalties, const float* presence_penalties,
    const float* frequency_penalties, const bool use_repetition, const bool use_presence, const bool use_frequency,
    const int** outputIds, const int* sequenceLengths, const int batchSize, const int vocabSize, int maxSeqLen,
    hipStream_t stream)
{
    TLLM_LOG_DEBUG("%s start", __PRETTY_FUNCTION__);
    dim3 block(min(maxSeqLen, 1024));
    dim3 grid(batchSize);
    // FIXME(nkorobov): with long sequences we might hit upper smem limit
    size_t smemSize = maxSeqLen * sizeof(int);
    if (use_repetition || use_presence)
    {
        smemSize += maxSeqLen * sizeof(float);
    }
    if (smemSize >= 46 * 1024)
    {
        /* Set 46KB threshold here because we have to take static/driver shared memory into consideration. */
        hipError_t res = hipFuncSetAttribute(reinterpret_cast<const void*>(
            batchApplyRepetitionPenalty<T>), hipFuncAttributeMaxDynamicSharedMemorySize, smemSize);
        TLLM_CHECK_WITH_INFO(res == hipSuccess,
            "Sequence Length is too long for the batchApplyRepetitionPenalty kernel (not enough shared memory).");
    }
    batchApplyRepetitionPenalty<T><<<grid, block, smemSize, stream>>>(logits, repetition_penalties, presence_penalties,
        frequency_penalties, use_repetition, use_presence, use_frequency, outputIds, sequenceLengths, batchSize,
        vocabSize, maxSeqLen);
}

template void invokeBatchApplyRepetitionPenalty(float* logits, const float* repetition_penalties,
    const float* presence_penalties, const float* frequency_penalties, const bool use_repetition,
    const bool use_presence, const bool use_frequency, const int** outputIds, const int* sequenceLengths,
    const int batchSize, const int vocabSize, int maxSeqLen, hipStream_t stream);

template void invokeBatchApplyRepetitionPenalty(half* logits, const float* repetition_penalties,
    const float* presence_penalties, const float* frequency_penalties, const bool use_repetition,
    const bool use_presence, const bool use_frequency, const int** outputIds, const int* sequenceLengths,
    const int batchSize, const int vocabSize, int maxSeqLen, hipStream_t stream);

template <typename T>
__global__ void batchApplyMinLengthPenalty(T* logits, const int* minLengths, const int* endIds,
    const int* sequenceLengths, const int* contextLengths, const int vocabSizePaddeded)
{
    int bid = threadIdx.x + blockIdx.x * blockDim.x; // batch index
    auto const contextLength{contextLengths == nullptr ? 0 : contextLengths[bid]};
    if (sequenceLengths[bid] - contextLength < minLengths[bid])
    {
        T maskVal = (std::is_same<T, half>::value) ? -65504.0f : -FLT_MAX;
        logits[bid * vocabSizePaddeded + endIds[bid]] = maskVal;
    }
}

template <typename T>
void invokeMinLengthPenalty(T* logits, const int* minLengths, const int* endIds, const int* sequneceLengths,
    const int* contextLengths, const int batchSize, const int vocabSizePaddeded, hipStream_t stream)

{
    const int blockSize = min(batchSize, 1024);
    const int gridSize = (batchSize + blockSize - 1) / blockSize;
    batchApplyMinLengthPenalty<<<gridSize, blockSize, 0, stream>>>(
        logits, minLengths, endIds, sequneceLengths, contextLengths, vocabSizePaddeded);
}

template void invokeMinLengthPenalty(float* logits, const int* minLengths, const int* endIds,
    const int* sequneceLengths, const int* contextLengths, const int batchSize, const int vocabSizePaddeded,
    hipStream_t stream);

template void invokeMinLengthPenalty(half* logits, const int* minLengths, const int* endIds, const int* sequneceLengths,
    const int* contextLengths, const int batchSize, const int vocabSizePaddeded, hipStream_t stream);

} // namespace kernels
} // namespace tensorrt_llm
