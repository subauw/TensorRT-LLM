#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/stopCriteriaKernels.h"

using namespace tensorrt_llm::common;

namespace tensorrt_llm
{
namespace kernels
{
__global__ void stopWordsCriterion(const int** outputIds, const int** parentIds, const int* stopWords,
    FinishedState* finished, const int* sequenceLengths, size_t stopWordsLen, int batchSize, int beamWidth,
    int maxSeqLen)
{
    const int id = blockIdx.x * blockDim.x + threadIdx.x;
    const int batchIdx = blockIdx.y / beamWidth;
    const int beamIdx = blockIdx.y % beamWidth;

    const int* baseStopWords = stopWords + batchIdx * 2 * stopWordsLen;
    const int* baseOffsets = baseStopWords + stopWordsLen;

    if (id >= stopWordsLen || baseOffsets[id] < 0)
    {
        return;
    }

    const int itemEnd = baseOffsets[id];
    const int itemStart = (id > 0) ? baseOffsets[id - 1] : 0;
    const int itemSize = itemEnd - itemStart;

    // The single-token case unconditionally bans the token
    bool shouldStop = false;

    // Need to minus 1 because the sequenceLengths is updated in this step
    const int currentStep = sequenceLengths[blockIdx.y] - 1;
    // Enough previously generated tokens to look for a match
    if (currentStep + 1 >= itemSize)
    {
        shouldStop = true;
        int parentId = beamIdx;
        const bool gatherBeam = beamWidth > 1;

        for (int tokenIdx = itemSize - 1; tokenIdx >= 0; tokenIdx--)
        {
            const int previousToken
                = outputIds[batchIdx][parentId * maxSeqLen + currentStep - (itemSize - 1) + tokenIdx];
            if (previousToken != baseStopWords[itemStart + tokenIdx])
            {
                shouldStop = false;
                break;
            }
            if (gatherBeam)
            {
                parentId = parentIds == nullptr
                    ? 0
                    : parentIds[batchIdx][parentId * maxSeqLen + currentStep - (itemSize - 1) + tokenIdx];

                if (parentId < 0 || parentId >= beamWidth)
                {
                    shouldStop = false;
                    break;
                }
            }
        }
    }

    if (shouldStop)
    {
        finished[batchIdx * beamWidth + beamIdx].setFinishedStopWords();
    }
}

void invokeStopWordsCriterion(const int** outputIds, const int** parentIds, const int* stopWords,
    FinishedState* finished, const int* sequenceLengths, size_t stopWordsLen, int batchSize, int beamWidth,
    int maxSeqLen, hipStream_t stream)
{
    // Check if we have sampled a word from the stopWords list. If so, stop the sequence.
    dim3 block, grid;
    constexpr size_t maxBlockSize{256};
    block.x = min(((stopWordsLen + 32 - 1) / 32) * 32, maxBlockSize);
    grid.x = (stopWordsLen + block.x - 1) / block.x;
    grid.y = batchSize * beamWidth;

    stopWordsCriterion<<<grid, block, 0, stream>>>(
        outputIds, parentIds, stopWords, finished, sequenceLengths, stopWordsLen, batchSize, beamWidth, maxSeqLen);
    sync_check_cuda_error();
}

__global__ void lengthCriterion(FinishedState* finished, int* finishedSum, const uint32_t* sequenceLimitLength,
    const int* sequenceLengths, int batchSize, int beamWidth)
{
    int threadFinishedCount = 0;
    for (int index = threadIdx.x; index < batchSize * beamWidth; index += blockDim.x)
    {
        const int batchIdx = index / beamWidth;

        auto finishState = finished[index];

        if (sequenceLengths[index] >= sequenceLimitLength[batchIdx])
        {
            finishState.setFinishedMaxLength();
        }
        threadFinishedCount += finishState.isFinished() ? 1 : 0;
        finished[index] = finishState;
    }

    if (finishedSum)
    {
        int blockFinishedCount = 0;
        if (blockDim.x <= 32)
        {
            blockFinishedCount = warpReduceSum(threadFinishedCount);
        }
        else
        {
            blockFinishedCount = blockReduceSum(threadFinishedCount);
        }
        __syncthreads();

        if (threadIdx.x == 0)
        {
            finishedSum[0] = blockFinishedCount;
        }
    }
}

void invokeLengthCriterion(FinishedState* finished, int* finishedSum, const uint32_t* sequenceLimitLength,
    const int* sequenceLengths, int batchSize, int beamWidth, hipStream_t stream)
{
    // Check if we have attained the sequence length limit. If so, stop the
    // sequence. In addition, check if all sequences are stopped and return the
    // result in shouldStop
    dim3 block{min(512, uint32_t(batchSize * beamWidth))};
    dim3 grid{1};

    lengthCriterion<<<grid, block, 0, stream>>>(
        finished, finishedSum, sequenceLimitLength, sequenceLengths, batchSize, beamWidth);
    sync_check_cuda_error();
}

} // namespace kernels
} // namespace tensorrt_llm
