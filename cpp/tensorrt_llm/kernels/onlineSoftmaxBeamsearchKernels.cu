/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/kernels/onlineSoftmaxBeamsearchKernels.h"

using namespace tensorrt_llm::common;

namespace tensorrt_llm
{
namespace kernels
{

template <typename T, int MAX_K>
void topK_softMax_kernelLauncher(const T* log_probs, const T* bias, const FinishedState* finished,
    const int* sequence_lengths, float* cum_log_probs, float* output_log_probs, int** output_ids_ptr,
    void* temp_storage, const int temp_storage_size, BeamHypotheses* beam_hyps, const int batch_size,
    const int beam_width, const int vocab_size, const int* end_ids, const float* diversity_rates,
    const float* length_penalties, hipStream_t stream);

#define CASE_K(MAX_K)                                                                                                  \
    topK_softMax_kernelLauncher<T, MAX_K>(log_probs, bias, finished, sequence_lengths, cum_log_probs,                  \
        output_log_probs, output_ids_ptr, temp_storage, temp_storage_size, beam_hyps, batch_size, beam_width,          \
        vocab_size, end_ids, diversity_rates, length_penalties, stream);                                               \
    break;

template <typename T>
void invokeTopkSoftMax(const T* log_probs, const T* bias, const FinishedState* finished, const int* sequence_lengths,
    float* cum_log_probs, float* output_log_probs, int** output_ids_ptr, void* temp_storage,
    const int temp_storage_size, BeamHypotheses* beam_hyps, const int batch_size, const int beam_width,
    const int vocab_size, const int* end_ids, const float* diversity_rates, const float* length_penalties,
    hipStream_t stream)
{
    int log_beam_width(0);
    int recursor(beam_width - 1);
    while (recursor >>= 1)
        ++log_beam_width;

    switch (log_beam_width)
    {
    // 0 < beam_width <= 4
    case 0:        // 1, 2
    case 1:        // 3, 4
        CASE_K(4)
    case 2:        // 4 < beam_width <= 8
        CASE_K(8)
#ifndef FAST_BUILD // For fast build, skip case 3, 4, 5
    case 3:        // 9 < beam_width <= 16
        CASE_K(16)
    case 4:        // 16 < beam_width <= 32
        CASE_K(32)
    case 5:        // 32 < beam_width <= 64
        CASE_K(64)
#endif             // FAST_BUILD
    default:
        throw std::runtime_error(
            fmtstr("%s:%d Topk kernel of beam search does not support beam_width=%d", __FILE__, __LINE__, beam_width));
    }
}

#undef CASE_K

template void invokeTopkSoftMax<float>(const float* log_probs, const float* bias, const FinishedState* finished,
    const int* sequence_lengths, float* cum_log_probs, float* output_log_probs, int** output_ids_ptr, void* tmp_storage,
    const int temp_storage_size, BeamHypotheses* beam_hyps, const int batch_size, const int beam_width,
    const int vocab_size, const int* end_ids, const float* diversity_rates, const float* length_penalties,
    hipStream_t stream);

template void invokeTopkSoftMax<half>(const half* log_probs, const half* bias, const FinishedState* finished,
    const int* sequence_lengths, float* cum_log_probs, float* output_log_probs, int** output_ids_ptr, void* tmp_storage,
    const int temp_storage_size, BeamHypotheses* beam_hyps, const int batch_size, const int beam_width,
    const int vocab_size, const int* end_ids, const float* diversity_rates, const float* length_penalties,
    hipStream_t stream);

} // namespace kernels
} // namespace tensorrt_llm
